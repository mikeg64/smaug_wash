  //Number threads per block
  int NTPB=512;
  //Num blocks is determined by size of zeropadded 2^n size array
  int numBlocks = (ndimp+NTPB-1) / NTPB;
  //Shared memory
  int smemSize = NTPB * sizeof(double);
  //Array to store maximum values for reduction in host memory 
  double *h_cmax = (double*)malloc(numBlocks*sizeof(double));

  hipMalloc((void**)&d_cmax, numBlocks*sizeof(double));
  //Array to store maximum values for reduction in GPU global memory 
  hipMalloc((void**)&d_bmax, numBlocks*sizeof(double)); 
  
  //set maximum value to zero and update values in GPU memory
  (*p)->cmax=0.0;
  hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 
  //copy speeds and temporary values to device memory 
  copytotemp_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wd, *d_wtemp,cfast);
  int i=0;

  //find the maximum in each block
  for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
  hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);

  reductionmax_parallel<<<numBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,ndimp);
  hipDeviceSynchronize();
  hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

  //compare the maxima for all of the blocks and determine maximum value
  for( i=0;i<numBlocks;i++)          		
                if(h_cmax[i]>((*p)->cmax)) ((*p)->cmax)=h_cmax[i];


 //determine maximum value
 hipMemcpy(*d_wtemp, ((*wd)+(soundspeed*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
