#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,xc3;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa;
   real s_period,s_period2,s_period3;
   real qt,tdep,tdepx,tdepy;

   real vx,vy,vz;

   real exp_x,exp_y,exp_z,exp_xyz;


   real xp,yp,zp;
   int i,j,k;
 	  
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

     xc1=0.5e6;
    xc2=1.27e6;
    xc3=1.27e6;

          xp=(p->xmin[1])+(((real)j)*(p->dx[1]))-xc2;
          zp=(p->xmin[0])+(((real)i)*(p->dx[0]))-xc1;
          yp=(p->xmin[2])+(((real)k)*(p->dx[2]))-xc3;
     // xx=x(ix_1,ix_2,ix_3,2)-xc2
     // yy=x(ix_1,ix_2,ix_3,3)-xc3
     // zz=x(ix_1,ix_2,ix_3,1)-xc1  


          xp=wd[fencode3_MODID(p,ii,pos2)]-xc2;
          zp=wd[fencode3_MODID(p,ii,pos1)]-xc1;
          yp=wd[fencode3_MODID(p,ii,pos3)]-xc3;  
     

    xxmax=2.54e6;
    yymax=2.54e6;

    //dx=p->dx[1];
    //dy=p->dx[2];
    //dz=p->dx[0];


    //dx=0.1e6;
    //dy=0.1e6;
    //dz=0.05e6;


    dx=0.5e6;
    dy=0.5e6;
    dz=0.25e6;
   


    aa=0.1;
    s_period=180.0;
    s_period2=300.0;
    s_period3=30.0;
    tdep=1.00;
    qt=p->qt;

        //exp_z=exp(-zz**2.d0/(delta_z**2.d0))
        //exp_x=exp(-xx**2.d0/(delta_x**2.d0))
        //exp_y=exp(-yy**2.d0/(delta_y**2.d0))       
        //exp_xyz=exp_x*exp_y*exp_z
        exp_z=exp(-zp*zp/(dz*dz));
        exp_x=exp(-xp*xp/(dx*dx));
        exp_y=exp(-yp*yp/(dy*dy));       
        exp_xyz=exp_x*exp_y*exp_z;

        //vvx(ix_1,ix_2,ix_3)=AA*yy/yymax*exp_xyz*tdep    
        //vvy(ix_1,ix_2,ix_3)=-AA*xx/xxmax*exp_xyz*tdep 

        //torsional driver
        //vx=(aa*yp/yymax)*exp_xyz*tdep;    
        //vy=-(aa*xp/xxmax)*exp_xyz*tdep;


       //washing machine driver
        tdepx=sin(qt*2.0*PI/s_period)+sin(qt*2.0*PI/s_period2)+sin(qt*2.0*PI/s_period3);
        tdepy=sin((qt-(0.25*s_period))*2.0*PI/s_period)+sin((qt-(0.25*s_period))*2.0*PI/s_period2)+sin((qt-(0.25*s_period3))*2.0*PI/s_period);
        aa=aa*exp(-(qt-120.5)/90); //amplitude decay after 582s
        vx=aa*exp_xyz*tdepx;    
        vy=aa*exp_xyz*tdepy;




 

                           w[fencode3_MODID(p,ii,mom2)]+=(p->dt)*vx*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
  
                           w[fencode3_MODID(p,ii,mom3)]+=(p->dt)*vy*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);

                          w[fencode3_MODID(p,ii,energy)]+=(p->dt)*(vx*vx+vy*vy)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;

       // if(i==9 && j==63 && k==63 && ((p->it)%1000)==0 )
        //if(i==9 && j>61 && j<64 && k>61 && k<64 )
        //if(i>0 && i<10 && j>60 && j<65 && k>60 && k<65 && ((p->it)%1000)==0) 
	//{
               // p->test=(w[fencode3_MODID(p,ii,energy)]);
               // p->chyp[0]=(w[fencode3_MODID(p,ii,mom2)]);
               // p->chyp[1]=(w[fencode3_MODID(p,ii,mom3)]);


        //       printf("%d %d %d %f %f %f %f %f %f\n",i,j,k,qt,tdepx,tdepy,exp_xyz,vx,vy); 
	//}

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {


}

