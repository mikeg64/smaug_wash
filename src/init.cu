#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/iotypes.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_i.cuh"
#include "../include/init_user_i.cuh"


//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));

   
 #ifdef USE_SAC_3D
   int kp;
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
		//b[i+j*(p->n[0])]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read

	    /*for(int f=0; f<NVAR; f++)
            { 		         
                          for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=0;
	    }*/


//	 __syncthreads();

			}

        	
	 __syncthreads();


    /* #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
     
               for(int f=vel1; f<NDERV; f++)
                    wd[fencode3_i(p,ii,f)]=0.0;
     

 __syncthreads(); */



     #ifdef USE_SAC_3D
      // if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[2]>1 && ii[0]<(p->n[0])-1 && ii[1]<(p->n[1])-1 && ii[2]<(p->n[2])-1)
         if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1])   && ii[2]<(p->n[2])) 
     #else
      // if((p->readini==0) && ii[0]>2 && ii[1]>2 && ii[0]<(p->n[0])-3 && ii[1]<(p->n[1])-3)  //this form for OZT test???? 
     
     
     //if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[0]<(p->n[0])-1 && ii[1]<(p->n[1])-1)  //this form for OZT test???? 
        if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1]))  //this form for BW test  //still issue here
     #endif
	{


            #ifdef ADIABHYDRO
		    if(i> (((p->n[0])/2)-2) && i<(((p->n[0])/2)+2) && j>(((p->n[1])/2)-2) && j<(((p->n[1])/2)+2) ) 
				;//w[fencode3_i(p,ii,rho)]=1.3;
            #else
                   // init_alftest (real *w, struct params *p,int i, int j)
                   // init_alftest(w,p,i,j);
                   // init_ozttest (real *w, struct params *p,int i, int j)
                   // init_ozttest(w,p,i,j);
                   // init_bwtest(w,p,i,j);

	           //default values for positions these may be updated by the initialisation routines
                   wd[fencode3_i(p,ii,delx1)]=(p->dx[0]);
		   wd[fencode3_i(p,ii,delx2)]=(p->dx[1]);
                   wd[fencode3_i(p,ii,pos1)]=(p->xmin[0])+ii[0]*(p->dx[0]);
		   wd[fencode3_i(p,ii,pos2)]=(p->xmin[1])+ii[1]*(p->dx[1]);
                 #ifdef USE_SAC_3D
		   wd[fencode3_i(p,ii,pos3)]=(p->xmin[2])+ii[2]*(p->dx[2]);
                   wd[fencode3_i(p,ii,delx3)]=(p->dx[2]);
                 #endif

                   //init_user_i(w,p,ii);  //initilise using w field

                   //commented out because spicule problem
                   //constructed on host
               if(p->mode!=3)
                   init_user_i(wmod,wd,p,ii);
           #endif

	

        }
	
	 __syncthreads();


       





     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
        /*for(int f=energyb; f<NVAR; f++)
             if(f != rhob)
                      w[fencode3_i(p,ii,f)]=0.0;*/
        //w[fencode3_i(p,ii,b2b)]=w[fencode3_i(p,ii,b3b)];
        for(int f=rho; f<NVAR; f++)
        {               
                  //wmod[fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];
                  //wmod[  (((3*(1+(p->rkon)))-1)*NVAR*dimp)+fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];              
                  dwn1[fencode3_i(p,ii,f)]=0;

                  //initial value of ord changed to 1 ensure have correct background fields set
                  for(ord=1;ord<(2+3*(p->rkon==1));ord++)
                  {
                              //only the wmod field is used w now redundant
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=wmod[fencode3_i(p,ii,f)];

                              //original version using w
                              //wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=w[fencode3_i(p,ii,f)];
                              //wmod[fencode3_i(p,ii,b2b)+ord*NVAR*dimp]=w[fencode3_i(p,ii,b3b)];
                  }
  /*        int dir=0;
 for(int field=rho;field<=rho ; field++)
if( ii[0]<4 && (p->ipe)==0  && ((p)->it)==1 && ( isnan(wmod[fencode3_i(p,ii,field)])|| wmod[fencode3_i(p,ii,field)]==0 ))
        { 
    				printf("nant %d %d %d %d %lg %lg \n",ii[0],ii[1],field,dir, wmod[fencode3_i(p,ii,rho)],wmod[fencode3_i(p,ii,field)] );
}*/
                  
        }

        for(int f=tmp1; f<NTEMP; f++)
                 wtemp[fencode3_i(p,ii,f)]=0;


	/*for(int field=rho;field<=rho ; field++)
	if(  (p->ipe)==0  && (  wmod[fencode3_i(p,ii,field)]==0 ))
		{ 
	    				printf("nanti %d %d %d %d %lg %lg \n",ii[0],ii[1],field,0, wmod[fencode3_i(p,ii,rho)],wmod[fencode3_i(p,ii,field)+dimp*NVAR] );
	}*/


}

 __syncthreads();



}

__global__ void updatemod_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));

   
 #ifdef USE_SAC_3D
   int kp;
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif






 
     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
        for(int f=rho; f<NVAR; f++)
        {               
                  for(ord=1;ord<(2+3*(p->rkon==1));ord++)
                  {
                              //wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=w[fencode3_i(p,ii,f)];
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=wmod[fencode3_i(p,ii,f)];

                            // if(p->ipe==0    && f==rho)
                            //    printf("wmod,w %d %d %lg %lg\n",ii[0],ii[1],wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp],w[fencode3_i(p,ii,f)]);
 
                  }
          int dir=0;


 //for(int field=rho;field<=rho ; field++)
//if( /*ii[0]<4 &&*/ (p->ipe)==0  && /*((p)->it)==1 &&*/ (/* isnan(wmod[fencode3_i(p,ii,field)])||*/ wmod[fencode3_i(p,ii,field)]==0 ))
//        { 
//    				printf("nant %d %d %d %d %lg %lg \n",ii[0],ii[1],field,dir, wmod[fencode3_i(p,ii,rho)],wmod[fencode3_i(p,ii,field)] );
//}
                  
        }




}

 __syncthreads();



}


 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
//set up a temporary grid

__global__ void gridsetup_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp,kp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));
   kp=0;
   
 #ifdef USE_SAC_3D
 
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


     #ifdef USE_SAC_3D
       if(ii[0]>0 && ii[0]<(p->n[0]-1) && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
     {


        switch(dir)
        {

    case 0:
           wtemp2[encode3p2_i(p,ip+1,jp+1,kp+1,tmpnui)]=wd[fencode3_i(p,ii,pos1)];
    break;
    case 1:
           wtemp2[encode3p2_i(p,ip+1,jp+1,kp+1,tmpnui1)]=wd[fencode3_i(p,ii,pos2)];
    break;
    #ifdef USE_SAC_3D
           case 2:
                        wtemp2[encode3p2_i(p,ip+1,jp+1,kp+1,tmpnui2)]=wd[fencode3_i(p,ii,pos3)];
           break;
     #endif
           }
     }


        	
	 __syncthreads();




       





}




 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
//calculate the dx values

__global__ void setupdx_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp,kp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));

   
 #ifdef USE_SAC_3D
 
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

   //calculate the dx values


	    switch(dir)
	    {
		     case 0:
		     #ifdef USE_SAC_3D
		       if( ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 &&  ii[1]<(p->n[1])+1 && ii[2]>0 &&  ii[2]<(p->n[2])+1)
		     #else
		       if( ii[0]>0 && ii[0]<(p->n[0])+1  && ii[1]>0 && ii[1]<(p->n[1])+1)
		     #endif
	                wd[fencode3_i(p,ii,delx1)]=0.5*(wtemp2[encode3p2_i(p,ip+1,jp,kp,tmpnui)]-wtemp2[encode3p2_i(p,ip-1,jp,kp,tmpnui)]);
		     break;
	
		     case 1:
		     #ifdef USE_SAC_3D
		       if(ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 &&  ii[1]<(p->n[1])+1 && ii[2]>0 &&  ii[2]<(p->n[2])+1)
		     #else
		       if(ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 && ii[1]<(p->n[1])+1)
		     #endif
			wd[fencode3_i(p,ii,delx2)]=0.5*(wtemp2[encode3p2_i(p,ip,jp+1,kp,tmpnui)]-wtemp2[encode3p2_i(p,ip,jp-1,kp,tmpnui)]);
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:

		       if(ii[0]>0 && ii[0]<(p->n[0])+1 && ii[1]>0 && ii[1]<(p->n[1])+1 && ii[2]>0 && ii[2]<(p->n[2])+1)
			wd[fencode3_i(p,ii,delx3)]=0.5*(wtemp2[encode3p2_i(p,ip,jp,kp+1,tmpnui)]-wtemp2[encode3p2_i(p,ip,jp,kp-1,tmpnui)]);
		     break;			
		     #endif
	     }
     
        	
	 __syncthreads();







       





}

 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
//intialise temporrary matrix needs t be completed
__global__ void zerotempv_parallel(struct params *p, real *w, real *wnew, real *wmod, 
real *dwn1,  real *wd, real *wtemp, real *wtemp1, real *wtemp2,  int dir)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  real maxt=0,max3=0, max1=0;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];


  
   int ip,jp;



  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
//int shift=order*NVAR*dimp;




//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {


        for(int f=d1; f<=d3; f++)
     #ifdef USE_SAC_3D
                 wtemp2[encode3p2_i(p,ii[0],ii[1],ii[2],tmpnui)]=0;
     #else
                 wtemp2[encode3p2_i(p,ii[0],ii[1],k,tmpnui)]=0;
     #endif

      if(i==((p->n[0])-1))
      {
        wtemp2[encode3p2_i(p,i+1,j,k,tmpnui)]=0;
        wtemp2[encode3p2_i(p,i+2,j,k,tmpnui)]=0;
      }
      if(j==((p->n[1])-1))
      {
          wtemp2[encode3p2_i(p,i,j+1,k,tmpnui)]=0;
          wtemp2[encode3p2_i(p,i,j+2,k,tmpnui)]=0;
      }

     #ifdef USE_SAC_3D
      if(k==((p->n[2])-1))
      {
          wtemp2[encode3p2_i(p,i,j,k+1,tmpnui)]=0;
          wtemp2[encode3p2_i(p,i,j,k+2,tmpnui)]=0;
      }

     #endif
      if(j==((p->n[1])-1)  && i==((p->n[0])-1))
      {
          for(int di=0; di<2; di++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_i(p,i+1+di,j+1+dj,k,tmpnui)]=0;
      }
     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1)  && k==((p->n[2])-1))
      {
          for(int di=0; di<2; di++)
             for(int dk=0; dk<2; dk++)
                   wtemp2[encode3p2_i(p,i+1+di,j,k+1+dk,tmpnui)]=0;
      }
      #endif

    

     #ifdef USE_SAC_3D
      if(j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_i(p,i,j+1+dj,k+1+dk,tmpnui)]=0;
      }
      #endif

     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1) && j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
               for(int di=0; di<2; di++)
                   wtemp2[encode3p2_i(p,i+1+di,j+1+dj,k+1+dk,tmpnui)]=0;
      }
      #endif

   }

}



/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
  
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  


}

int cusync(struct params **p)
{

  #ifdef USE_GPUD
     
         for(int igid=0; igid<((*p)->npe); igid++)
         {
                (*p)->ipe=igid;
                hipSetDevice((*p)->gpid[igid]) ;
                
  #endif
  hipDeviceSynchronize();
  #ifdef USE_GPUD
                 (*p)->ipe=0;
                 hipSetDevice((*p)->gpid[0]) ;
          }
  #endif
  return 0;
}

int cusetgpu(struct params **p)
{
  #ifdef USE_GPUD
    if(((*p)->ipe)==-1)
    {
         for(int igid=0; igid<((*p)->npe); igid++)
                (*p)->gpid[igid]=igid ;
    }
    else
      hipSetDevice((*p)->gpid[(*p)->ipe]) ;
                
  #endif
 
  return 0;
}

int cuinit(struct params **p, struct bparams **bp, real **wmod,real **wnew, real **wd, struct state **state, struct params **d_p, struct bparams **d_bp, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  int dir;
 /* hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }

  #ifdef USE_MPI
     int lipe=(*p)->ipe;
     int gpugid=lipe/4;
     selectedDevice=lipe-4*gpugid;
  #endif
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }


        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, selectedDevice);
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        hipSetDevice(selectedDevice) ;
        printf("> gpuDeviceInit() CUDA device [%d]: %s %s\n", selectedDevice, deviceProp.name, getenv("HOSTNAME"));



  hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");*/
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct bparams *adbp;
  struct state *ads;


 
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  
if((*p)->mode != 3)
{
	if(((*p)->rkon)==1)
	  hipMalloc((void**)d_wmod, 6*NVAR*dimp*sizeof(real));
	else
	  hipMalloc((void**)d_wmod, 3*NVAR*dimp*sizeof(real));

	  hipMalloc((void**)d_dwn1, NVAR*dimp*sizeof(real));
	  hipMalloc((void**)d_wd, NDERV*dimp*sizeof(real));
	  hipMalloc((void**)d_wtemp, NTEMP*dimp*sizeof(real));


	  #ifdef USE_SAC
	  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)*sizeof(real));
	  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
	  #endif
	  #ifdef USE_SAC_3D
	  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)* (((*p)->n[2])+1)*sizeof(real));
	  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real));
	  #endif

	  //hipMalloc((void**)&adw, NVAR*dimp*sizeof(real));
	  //hipMalloc((void**)&adwnew, NVAR*dimp*sizeof(real));

	  hipMalloc((void**)&adbp, sizeof(struct bparams));
	  hipMalloc((void**)&adp, sizeof(struct params));
	  hipMalloc((void**)&ads, sizeof(struct state));
	 // checkErrors_i("memory allocation");

	printf("ni is %d\n",(*p)->n[1]);

	   // *d_b=adb;
	    *d_bp=adbp;
	    *d_p=adp;
	    //*d_w=adw;
	    //*d_wnew=adwnew;
	    *d_state=ads;

	     
	//printf("allocating %d %d %d %d\n",dimp,(*p)->n[0],(*p)->n[1],(*p)->n[2]);
	printf("allocating %d %d %d \n",dimp,(*p)->n[0],(*p)->n[1]);




	//printf("here1\n");






	 
	    //printf("here2\n");

	    //hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
	    hipMemcpy(*d_wmod, *wmod, 2*(1+(((*p)->rkon)==1))*NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
	    hipMemcpy(*d_wd, *wd, NDERV*dimp*sizeof(real), hipMemcpyHostToDevice);






	//printf("here3\n");






	   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
	   // printf("here\n");
	    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
	    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
	    
	    dim3 dimBlock(16, 1);
	    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
	    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
	   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
	   

	    printf("calling initialiser\n");
	     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
	    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
	     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2);


}//end of if(p->mode !=3)

     //hipDeviceSynchronize();
     

//checkErrors_i("memory allocation");

     //copy data back to cpu so we can compute and update the grid (on the cpu)

 
    //hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

/*if((*p)->mode==3)
{
  
  int ii[3];
  ii[0]=0;
  ii[1]=0;
  ii[2]=0;
 init_user_i(*wmod,*wd,*p,ii);
}*/







    //hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
    //setup the grid and dx values here


    //hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);


 //initialise grid on the gpu
 //we currently don't do this to avoid use of additional memory on GPU
 /*for(dir=0; dir<NDIM; dir++)
 {
     zerotempv_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2,dir);
     hipDeviceSynchronize();     
     gridsetup_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2,dir);
     hipDeviceSynchronize();
     setupdx_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2,dir);
     hipDeviceSynchronize();
  }*/

	    printf("called initialiser\n");
	//hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
if((*p)->mode !=3)
{
	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
}


//checkErrors_i("memory allocation");checkErrors_i("memory allocation");



	//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

        // printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->n[0])+((*p)->n[0])*((*p)->n[1])*b1]));



  return 0;



}




int cuupdatemod(struct params **p, struct bparams **bp,real **w, real **wnew, real **wd, struct state **state, struct params **d_p, struct bparams **d_bp,real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{
  int deviceCount;
  int dir;
 
  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct bparams *adbp;
  struct state *ads;

 
 
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   
    printf("calling updatemod\n");
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     updatemod_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2);
     //hipDeviceSynchronize();
     


 


  return 0;
}




/*! Cartesian or polar grid. Determine x at the boundaries.
! Determine often needed combinations of x, such as dx or dvolume.
! Determine variables for axial symmetry
!
! ixe          - edge coordinate of the grid touching the boundary region
! ixf          - coordinate inside of ixe
! qx           - x with an extended index range for calculation of dx   */

int initgrid(struct params **p,   struct state **state, real **wd, struct params **d_p,  real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{
    real *ttemp2;
    int ii[NDIM];
    int ii1[3],ii2[3],ix;
    int ip,jp,kp,kpo;
    int dir,dir1,dir2;
    int ixmin,ixmax,ixe,ixf;
    real *wda=*wd;
    //real *wa=*wmod;
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));

/*if(((*p)->ipe)==2)
      {
checkErrors_i("initgrid memory allocation");
}*/
 #ifdef USE_SAC_3D
 
   dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
    kp=0;
    //printf("called initgrid %d\n",(*p)->ipe);
    

    for(int i=0;i<3;i++)
    {
       ii1[i]=0;
       ii2[i]=0;
    }
    #ifdef USE_SAC
    ttemp2=(real *) malloc( (NTEMP2+2)*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
    #endif
    #ifdef USE_SAC_3D
    ttemp2=(real *)malloc((NTEMP2+2)*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real));
    #endif
    
     //hipMemcpy(*wmod, *d_wmod, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
     hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
     for(dir=0;dir<NDIM;dir++)
     for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
     		     #ifdef USE_SAC_3D
                   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
                 #endif
                 {
                        ip=ii[0];
                        jp=ii[1];
         		     #ifdef USE_SAC_3D
                       kp=ii[2];
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	                ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui)]= 0;
		     break;
	
		     case 1:
			 ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui1)]= 0;
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			 ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui2)]= 0;
		     break;			
		     #endif
	     }
      }	
 

     kp=1;
     kpo=0;
     for(dir=0;dir<NDIM;dir++)
        for(ii[0]=1; ii[0]<((*p)->n[0])+1; ii[0]++)
           for(ii[1]=1; ii[1]<((*p)->n[1])+1; ii[1]++)
		#ifdef USE_SAC_3D
		   for(ii[2]=1; ii[2]<((*p)->n[2])+1; ii[2]++)
		#endif
                {
                        ip=ii[0];
                        jp=ii[1];
         		     #ifdef USE_SAC_3D
                       kp=ii[2];
                       kpo=kp;
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	                ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui)]= (wda[encode3_i(*p,ip-1,jp-1,kp-1,pos1)]);
		     break;
	
		     case 1:
			 ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui1)]= (wda[(encode3_i(*p,ip-1,jp-1,kp-1,pos2))]);
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			 ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui2)]= (wda[(encode3_i(*p,ip-1,jp-1,kp-1,pos3))]);
		     break;			
		     #endif
	     }


      }	


   /* if((*p)->ipe==3   ) 
        for(ii[1]=1; ii[1]<((*p)->n[1])+1; ii[1]++)                                                            
        for(ii[0]=1; ii[0]<((*p)->n[0])+1; ii[0]++)    
           {
                      ip=ii[0];
                        jp=ii[1];
                         printf("ii0, ii1 %d %d %16.20f %16.20f\n",ip,jp, ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui)],ttemp2[(encode3p2_i(*p,ip,jp,kpo,tmpnui1))]);

            }*/


  	
   	//update grid edges
     kp=0;
     for(dir=0;dir<NDIM;dir++)
     {
                
                       
	    switch(dir)
	    {
		     case 0:
                       ixmax=((*p)->n[0])+1;//ixGmax1+1; 
                       ixmin=((*p)->n[0])-1;//ixmin1=ixGmax1+1                      

                      #ifdef USE_MULTIGPU
			if(((*p)->fullgridini)==1    ||   ((*p)->mpiupperb[dir])==1) ixmin=((*p)->n[0])+1;//ixGmax1+1;
                      #endif

                       ixe=ixmin-1; 
                       ixf=ixe-1;


                       //upper layers
			     for(dir1=0;dir1<NDIM;dir1++)
			     {
				     for(ii[0]=ixmin; ii[0]<=ixmax; ii[0]++)
				     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
				                        ix=ii[0];
                                                        ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[0]=ixe;
                                                       ii2[0]=ixf; 

 

                                                       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						      //ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (wda[fencode3_i(*p,ii1,pos1+dir1)]);
						  }

				}



                      //lower layers

                       ixmin=0;//ixmin1=ixGmin1-1;
                       ixmax=2;//ixmax1=ixGmin1-1 

                     #ifdef USE_MULTIGPU
			if(((*p)->fullgridini)==1    ||  ((*p)->mpilowerb[dir])==1) ixmax=0;
                      #endif

                  
                       ixe=ixmax+1; 
                       ixf=ixe+1;

			     for(dir1=0;dir1<NDIM;dir1++)
			     {
				     for(ii[0]=ixmin; ii[0]<=ixmax; ii[0]++)
				     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
							ix=ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[0]=ixe;
                                                       ii2[0]=ixf;




 
    ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
// ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]= (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])+ (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
   // qx(ix,ixmin2:ixmax2,jdim)=(1+abs(ixe-ix))*qx(ixe,ixmin2:ixmax2,jdim)- abs(ixe-ix) *qx(ixf,ixmin2:ixmax2,jdim)


//		if((*p)->ipe==0   && ii[1]==0) 
//                         printf("ii0, ii1 %d %d %16.20f %16.20f %d %d %d %d %d  %d %d %d %d\n",ip,jp, ttemp2[encode3p2_i(*p,ip,jp,kpo,tmpnui)],ttemp2[(encode3p2_i(*p,ip,jp,kpo,tmpnui1))],dir1,ixe,ixf,ixmin,ixmax,ii1[0],ii1[1],ii2[0],ii2[1]);





						  }

				}
		     break;
	
		     case 1:
                       ixmax=((*p)->n[1])+1;//ixGmax1+1; 
                       ixmin=((*p)->n[1])-1;//ixmin1=ixGmax1+1                      

                      #ifdef USE_MULTIGPU
			if(((*p)->fullgridini)==1    ||  ((*p)->mpiupperb[dir])==1) ixmin=((*p)->n[1])+1;//ixGmax1+1;
                      #endif
                     
                       ixe=ixmin-1; 
                       ixf=ixe-1;


                       //upper layers
			     for(dir1=0;dir1<NDIM;dir1++)
			     {
                 for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
				     for(ii[1]=ixmin; ii[1]<=ixmax; ii[1]++)
				     
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							ix=jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[1]=ixe;
                                                       ii2[1]=ixf; 





						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						      //ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (wda[fencode3_i(*p,ii1,pos1+dir1)]);
						  }

				}



                      //lower layers

                       ixmin=0;//ixmin1=ixGmin1-1;
                       ixmax=2;//ixmax1=ixGmin1-1 

                     #ifdef USE_MULTIGPU
			if(((*p)->fullgridini)==1    ||  ((*p)->mpilowerb[dir])==1) ixmax=0;
                      #endif
                
                       ixe=ixmax+1; 
                       ixf=ixe+1;

			     for(dir1=0;dir1<NDIM;dir1++)
			     {
			         for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)	
				     for(ii[1]=ixmin; ii[1]<=ixmax; ii[1]++)
				     		 #ifdef USE_SAC_3D
						   for(ii[2]=0; ii[2]<((*p)->n[2])+2; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							ix=jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[1]=ixe;
                                                       ii2[1]=ixf; 



                                                        

						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
//write(*,*) jdim,ixe,ixf,ix,ixmin1,ixmax1,ixmin2,ixmax2, qx(ixmin1:ixmax1,ix,jdim),qx(ixmin1:ixmax1,&
//                   ixe,jdim),qx(ixmin1:ixmax1,ixf,jdim)
//if((*p)->ipe==0   && ii[0]==0)
//                                                             printf("ixe, ix %d %d %d %d %d %d %16.20f %16.20f %16.20f\n",dir1,ixe,ixf,ix,ixmin,ixmax, ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)],ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))],ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
//if((*p)->ipe==0   && ii[0]==0)
//                                                             printf("ixe, ix %d %d %d %d %d %d %16.20f %16.20f %16.20f\n",dir1,ixe,ixf,ix,ixmin,ixmax, ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)],ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))],ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);


						  }

				}




		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
                       ixmax=((*p)->n[2])+1;//ixGmax1+1; 
                       ixmin=((*p)->n[2])-1;//ixmin1=ixGmax1+1                      

                      #ifdef USE_MULTIGPU
			if(((*p)->mpiupperb[dir])==1) ixmin=((*p)->n[2])+1;//ixGmax1+1;
                      #endif
                  
                       ixe=ixmin-1; 
                       ixf=ixe-1;


                       //upper layers
			     for(dir1=0;dir1<NDIM;dir1++)
			     {
                 for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
                 for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
				     
				     		 #ifdef USE_SAC_3D
						  
			        for(ii[2]=ixmin; ii[2]<=ixmax; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       ix=kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[2]=ixe;
                                                       ii2[2]=ixf; 
						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						      //ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (wda[fencode3_i(*p,ii1,pos1+dir1)]);
						  }

				}



                      //lower layers
                      //lower layers

                       ixmin=0;//ixmin1=ixGmin1-1;
                       ixmax=2;//ixmax1=ixGmin1-1 

                     #ifdef USE_MULTIGPU
			if(((*p)->fullgridini)==1    ||  ((*p)->mpilowerb[dir])==1) ixmax=0;
                      #endif
                   
                       ixe=ixmax+1; 
                       ixf=ixe+1;

			     for(dir1=0;dir1<NDIM;dir1++)
			     {
			         for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
                     for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)	
				     
				     		 #ifdef USE_SAC_3D
						   
						    for(ii[2]=ixmin; ii[2]<=ixmax; ii[2]++)
						 #endif
						 {
							ip=ii[0];
							jp=ii[1];
					 		     #ifdef USE_SAC_3D
						       ix=kp=ii[2];
						     #endif  
                                                       for(dir2=0;dir2<NDIM;dir2++)
                                                       {
                                                         ii1[dir2]=ii[dir2];
                                                         ii2[dir2]=ii[dir2];
                                                       }
                                                       ii1[2]=ixe;
                                                       ii2[2]=ixf; 
						       ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui+dir1)]=(1+abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii1,tmpnui+dir1))])-(abs(ixe-ix))* (ttemp2[(fencode3p2_i(*p,ii2,tmpnui+dir1))]);
						  }

				}



		     break;			
		     #endif
	     }
      }	


kp=0;

     for(dir=0;dir<NDIM;dir++)
        for(ii[0]=0; ii[0]<((*p)->n[0]); ii[0]++)
           for(ii[1]=0; ii[1]<((*p)->n[1]); ii[1]++)
		#ifdef USE_SAC_3D
		   for(ii[2]=0; ii[2]<((*p)->n[2]); ii[2]++)
		#endif
                {
                        ip=ii[0]+1;
                        jp=ii[1]+1;
         		     #ifdef USE_SAC_3D
                       kp=ii[2]+1;
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	                 (wda[fencode3_i(*p,ii,pos1)])=ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui)];
                      //  if(ip==1)
                      //  printf("delx 0 %d %d %16.20f  %16.20f \n",ii[0],ii[1],wda[(encode3_i(*p,ip-1,jp-1,kp,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp,delx2))]);
		     break;
	
		     case 1:
			  (wda[(fencode3_i(*p,ii,pos2))])=ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui1)];
//if(ip==1)
                 //       printf("delx 1 %d %d %16.20f  %16.20f \n",ii[0],ii[1],wda[(encode3_i(*p,ip-1,jp-1,kp,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp,delx2))]);

		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			  (wda[(fencode3_i(*p,ii,pos3))])=ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui2)];
		     break;			
		     #endif
	     }
      }	



   	//calculate dx
  kp=0;
  kpo=0;

                   for(dir=0;dir<NDIM;dir++)
                 {

  for(ii[0]=1; ii[0]<((*p)->n[0])+1; ii[0]++)
     for(ii[1]=1; ii[1]<((*p)->n[1])+1; ii[1]++)
  //for(ii[0]=0; ii[0]<((*p)->n[0]); ii[0]++)
  //   for(ii[1]=0; ii[1]<((*p)->n[1]); ii[1]++)

     		     #ifdef USE_SAC_3D
                   for(ii[2]=1; ii[2]<((*p)->n[2])+1; ii[2]++)
                 #endif
{

                        ip=ii[0];
                        jp=ii[1];
         		     #ifdef USE_SAC_3D
                       
                       kp=ii[2];
                        kpo=kp-1;
                     #endif                   
                       
	    switch(dir)
	    {
		     case 0:
	               // (wda[(encode3_i(*p,ip-1,jp-1,kpo,delx1))])=/*(*p)->dx[0];//*/0.5*(ttemp2[encode3p2_i(*p,ip+1,jp,kp,tmpnui)]-ttemp2[encode3p2_i(*p,ip-1,jp,kp,tmpnui)]);
                  (wda[(encode3_i(*p,ip-1,jp-1,kpo,delx1))])=/*(*p)->dx[0];//*/0.5*(ttemp2[encode3p2_i(*p,ip+1,jp,kp,tmpnui)]-ttemp2[encode3p2_i(*p,ip-1,jp,kp,tmpnui)]);

		if(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx1))]==0) wda[(encode3_i(*p,ip-1,jp-1,kpo,delx1))]=(*p)->dx[0];
	              //  if(ip==128  && jp==128 && kp==128)
                      //  printf("delx 0 %d %d %d %16.20f  %16.20f   %16.20f \n",ii[0]-1,ii[1]-1,ii[2]-1,wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx2))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx3))]);
		     break;
	
		     case 1:
			(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx2))])=/*(*p)->dx[1];//*/0.5*(ttemp2[encode3p2_i(*p,ip,jp+1,kp,tmpnui1)]-ttemp2[encode3p2_i(*p,ip,jp-1,kp,tmpnui1)]);
		if(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx2))]==0) wda[(encode3_i(*p,ip-1,jp-1,kpo,delx2))]=(*p)->dx[1];
	               // if(ip==128  && jp==128 && kp==128)
                       //   printf("delx 1 %d %d %d %16.20f  %16.20f   %16.20f \n",ii[0]-1,ii[1]-1,ii[2]-1,wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx2))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx3))]);

		        //printf("delx2 %d %d %g ",ii[0],ii[1],wda[(fencode3_i(*p,ii,delx2))]);
		     break;
		         
		     #ifdef USE_SAC_3D
		     case 2:
			(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx3))])=0.5*(ttemp2[encode3p2_i(*p,ip,jp,kp+1,tmpnui2)]-ttemp2[encode3p2_i(*p,ip,jp,kp-1,tmpnui2)]);
		if(wda[(encode3_i(*p,ip-1,jp-1,kpo,delx3))]==0) wda[(encode3_i(*p,ip-1,jp-1,kpo,delx3))]=(*p)->dx[2];
	              //  if(ip==128  && jp==128 && kp==128)
                      //  printf("delx 2 %d %d %d %16.20f  %16.20f   %16.20f \n",ii[0]-1,ii[1]-1,ii[2]-1,wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx1))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx2))],wda[(encode3_i(*p,ip-1,jp-1,kp-1,delx3))]);

		     break;			
		     #endif
	     }
      }
  printf("\n");
}


printf("dx=%g dy=%g\n",(*p)->dx[0], (*p)->dx[1] );




     kp=0;

   // if((*p)->ipe==3)
    // for(dir=0;dir<NDIM;dir++)
//for(ii[1]=0; ii[1]<((*p)->n[1])+2; ii[1]++)
        //for(ii[0]=0; ii[0]<((*p)->n[0])+2; ii[0]++)
           
            // {

              //          ip=ii[0];
              //          jp=ii[1];
                //if(ii[0]==0)
              //  printf("delx %d %d %16.20f  %16.20f  %16.20f  %16.20f \n",ii[0],ii[1],wda[(fencode3_i(*p,ii,pos1))],wda[(fencode3_i(*p,ii,pos2))],wda[(fencode3_i(*p,ii,delx1))],wda[(fencode3_i(*p,ii,delx2))]);
//printf("ttemp2 %d %d %16.20f  %16.20f  \n",ii[0],ii[1],ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui)],ttemp2[encode3p2_i(*p,ip,jp,kp,tmpnui1)]);
            //  }

  //  hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
  //  hipMemcpy(*d_wd, *wd, NDERV*dimp*sizeof(real), hipMemcpyHostToDevice);

 free(ttemp2);

 

    //hipMemcpy(*d_wmod, *wmod, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(*d_wd, *wd, NDERV*dimp*sizeof(real), hipMemcpyHostToDevice);


    //  
     



  

   
  return 0;



}


