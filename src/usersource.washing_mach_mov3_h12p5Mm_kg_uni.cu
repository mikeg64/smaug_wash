#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,xc3;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa;
   real s_period;
   real qt,tdep,tdepx,tdepy;

   real vx,vy,vz;
   real mvx, mvy, mvz;

   real exp_x,exp_y,exp_z,exp_xyz;
   real xc2_i,xc2_0,xc3_i,xc3_0;
   real tdec,t0;

   real xp,yp,zp;
   int i,j,k;
 	  
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

   qt=p->qt;
    aa=0.1;
    //aa=0.01;
    s_period=180.0;
    tdep=1.00;

   xc2_i=0.3e6;

   // xc2_i=0.0;
    xc3_i=1.55e6;

    //xc2_i=1.0e6;
    //xc3_i=1.0e6;




    xc2_0=1.25e6;
    xc3_0=1.25e6;
    t0=s_period;


     xc1=0.5e6;
    xc2=1.27e6;
    xc3=1.27e6;

    xc2=xc2_0+(qt-t0)*(xc2_0-xc2_i)/t0;
    xc3=xc3_i+(qt-t0)*(xc3_0-xc3_i)/t0;

        //  xp=(p->xmin[1])+(((real)j)*(p->dx[1]))-xc2;
        //  zp=(p->xmin[0])+(((real)i)*(p->dx[0]))-xc1;
        //  yp=(p->xmin[2])+(((real)k)*(p->dx[2]))-xc3;
     // xx=x(ix_1,ix_2,ix_3,2)-xc2
     // yy=x(ix_1,ix_2,ix_3,3)-xc3
     // zz=x(ix_1,ix_2,ix_3,1)-xc1  


          xp=wd[fencode3_MODID(p,ii,pos2)]-xc2;
          zp=wd[fencode3_MODID(p,ii,pos1)]-xc1;
          yp=wd[fencode3_MODID(p,ii,pos3)]-xc3;  
     

    //xc2=xc2_0+(qt-t0)*(xc2_0-xc2_i)/t0;
    //xc3=xc3_i+(qt-t0)*(xc3_0-xc3_i)/t0;


    xxmax=2.54e6;
    yymax=2.54e6;

    //dx=p->dx[1];
    //dy=p->dx[2];
    //dz=p->dx[0];


    //dx=0.1e6;
    //dy=0.1e6;
    //dz=0.05e6;


    dx=0.125e6;
    dy=0.125e6;
    dz=0.125e6;
   


    
    

        //exp_z=exp(-zz**2.d0/(delta_z**2.d0))
        //exp_x=exp(-xx**2.d0/(delta_x**2.d0))
        //exp_y=exp(-yy**2.d0/(delta_y**2.d0))       
        //exp_xyz=exp_x*exp_y*exp_z
        exp_z=exp(-zp*zp/(dz*dz));
        exp_x=exp(-xp*xp/(dx*dx));
        exp_y=exp(-yp*yp/(dy*dy));       
        exp_xyz=exp_x*exp_y*exp_z;

        //vvx(ix_1,ix_2,ix_3)=AA*yy/yymax*exp_xyz*tdep    
        //vvy(ix_1,ix_2,ix_3)=-AA*xx/xxmax*exp_xyz*tdep 

        //torsional driver
        //vx=(aa*yp/yymax)*exp_xyz*tdep;    
        //vy=-(aa*xp/xxmax)*exp_xyz*tdep;


       //washing machine driver
       // tdepx=sin(qt*2.0*PI/s_period);
       // tdepy=sin((qt-(0.25*s_period))*2.0*PI/s_period);
       // tdec=(qt<t0?exp(-(qt-t0)*(qt-t0)/(t0*t0)):1);
        //aa=aa*exp(-(qt-582)/90); //amplitude decay after 582s
        aa=aa*tdep*exp_xyz;
       // vx=aa*exp_xyz*tdepx;    
      //  vy=aa*exp_xyz*tdepy;


       mvx=w[fencode3_MODID(p,ii,mom1)];
       mvy=w[fencode3_MODID(p,ii,mom2)];
       mvz=w[fencode3_MODID(p,ii,mom3)];

 	w[fencode3_MODID(p,ii,rho)]+=(p->dt)*aa*w[fencode3_MODID(p,ii,rhob)];
       w[fencode3_MODID(p,ii,energy)]+=0.5*(p->dt)*(mvx*mvx+mvy*mvy+mvz*mvz)/(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);


                        //   w[fencode3_MODID(p,ii,mom2)]+=(p->dt)*vx*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
  
                         //  w[fencode3_MODID(p,ii,mom3)]+=(p->dt)*vy*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);

                        //  w[fencode3_MODID(p,ii,energy)]+=(p->dt)*(vx*vx+vy*vy)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;

        //if(i==9  && k==64 )  
        // if(i==9 && j==63 && k==63  )      
       // if(i==9 && j==63 && k==63 && ((p->it)%1000)==0 )
       // if(i==9 && j>61 && j<64 && k>61 && k<64 )
      //  if(i>0 && i<10 && k>60 && k<65 && j>=0 && j<15 && ((p->it)%10)==0) 
	//{
               // p->test=(w[fencode3_MODID(p,ii,energy)]);
               // p->chyp[0]=(w[fencode3_MODID(p,ii,mom2)]);
               // p->chyp[1]=(w[fencode3_MODID(p,ii,mom3)]);


              // printf("%d %d %d %d %f %f %f %f %f %f\n",p->it,i,j,k,qt,tdepx,tdepy,exp_xyz,vx,vy); 
             //printf("source %d %d %d %d %f %f %f %f  %f %f\n",p->it,i,j,k,qt,aa,exp_xyz,tdep,xc2,xc3); 

	//}

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {


}

