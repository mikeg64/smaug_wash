#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,xc3;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa, av;
   real s_period;
   real qt,tdep,tdepx,tdepy;

   real vx,vy,vz;
   real mvx, mvy, mvz;

   real exp_x,exp_y,exp_z,exp_xyz;
   real xc2_i,xc2_0,xc3_i,xc3_0;
   real tdec,t0;

   real xp,yp,zp;
   int i,j,k;
 	  
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

   qt=p->qt;
   //aa=0.005;
   //av=50.0;
   aa=0.01;
   av=10000.0;
 

   //xc2_i=0.6e6;  //y-dist
   xc2_i=1.0e6;  //y-dist

   // xc2_i=0.0;
  //  xc3_i=1.0e6;  //x-dist
   xc3_i=1.0e6;  //x-dist

    //xc2_i=1.0e6;
    //xc3_i=1.0e6;




    xc2_0=1.25e6;
    xc3_0=1.25e6;
 

     xc1=0.5e6;
    xc2=xc2_i;
    xc3=xc3_i;

 


          xp=wd[fencode3_MODID(p,ii,pos2)]-xc2;
          zp=wd[fencode3_MODID(p,ii,pos1)]-xc1;
          yp=wd[fencode3_MODID(p,ii,pos3)]-xc3;  
     

    xxmax=2.54e6;
    yymax=2.54e6;

    //dx=p->dx[1];
    //dy=p->dx[2];
    //dz=p->dx[0];


    //dx=0.1e6;
    //dy=0.1e6;
    //dz=0.05e6;


    //dx=0.125e6;
    //dy=0.125e6;
   // dz=0.125e6;
   

    dx=0.0625e6;
    dy=0.0625e6;
    dz=0.0625e6;

    
    

        //exp_z=exp(-zz**2.d0/(delta_z**2.d0))
        //exp_x=exp(-xx**2.d0/(delta_x**2.d0))
        //exp_y=exp(-yy**2.d0/(delta_y**2.d0))       
        //exp_xyz=exp_x*exp_y*exp_z
        exp_z=exp(-zp*zp/(dz*dz));
        exp_x=exp(-xp*xp/(dx*dx));
        exp_y=exp(-yp*yp/(dy*dy));       
        exp_xyz=exp_x*exp_y*exp_z;



       //washing machine driver
        aa=aa*exp_xyz;
        av=av*exp_xyz;
       // vx=aa*exp_xyz*tdepx;    
      //  vy=aa*exp_xyz*tdepy;

        w[fencode3_MODID(p,ii,rho)]+=aa*w[fencode3_MODID(p,ii,rhob)];
       w[fencode3_MODID(p,ii,mom2)]+=av*(w[fencode3_MODID(p,ii,rhob)]+w[fencode3_MODID(p,ii,rho)]);

       mvx=w[fencode3_MODID(p,ii,mom1)];
       mvy=w[fencode3_MODID(p,ii,mom2)];
       mvz=w[fencode3_MODID(p,ii,mom3)];

 	
       w[fencode3_MODID(p,ii,energy)]+=0.5*(mvx*mvx+mvy*mvy+mvz*mvz)/(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);

       // if(i==9 && j==63 && k==63 && ((p->it)%1000)==0 )
        //if(i==9 && j>61 && j<64 && k>61 && k<64 )
      //  if(i>0 && i<10 && k>60 && k<65 && j>=0 && j<15 && ((p->it)%10)==0) 
	//{
               // p->test=(w[fencode3_MODID(p,ii,energy)]);
               // p->chyp[0]=(w[fencode3_MODID(p,ii,mom2)]);
               // p->chyp[1]=(w[fencode3_MODID(p,ii,mom3)]);


              // printf("%d %d %d %d %f %f %f %f %f %f\n",p->it,i,j,k,qt,tdepx,tdepy,exp_xyz,vx,vy); 
            // printf("%d %d %d %d %f %f %f %f \n",p->it,i,j,k,qt,aa,exp_xyz,tdep); 

	//}

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {


}

