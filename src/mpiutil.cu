#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/iotypes.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_mpiu.cuh"
//#include "../include/init_user_mpiu.cuh"





__device__ __host__
int encodempiw (struct params *p,int ix, int iy, int iz, int field,int bound,int dim) {
  #ifdef USE_SAC_3D
    return (dim*(    4*NVAR*(         ((p->n[0])*(p->n[1]))+((p->n[1])*(p->n[2]))+((p->n[0])*(p->n[2]))   )           )+4*field*(         ((p->n[0])*(p->n[1]))+((p->n[1])*(p->n[2]))+((p->n[0])*(p->n[2]))   )+
bound*(         (dim==2)*((p->n[0])*(p->n[1]))   +  (dim==0)*((p->n[1])*(p->n[2]))  +   (dim==1)*((p->n[0])*(p->n[2]))    )+   (  (ix+iz*(p->n[0]))*(dim==1)+(iy+iz*(p->n[1]))*(dim==0)+(iz+ix*(p->n[2]))*(dim==2)    ));
  #else
    return (dim*(4*NVAR*((p->n[0])+(p->n[1])))+4*field*((p->n[0])+(p->n[1]))+bound*((dim==1)*(p->n[0])+(dim==0)*(p->n[1]))  +   (ix*(dim==1)+iy*(dim==0)));
  #endif
}

__device__ __host__
int encodempiw0 (struct params *p,int ix, int iy, int iz, int field,int bound) {
  #ifdef USE_SAC_3D
    return (4*field*(         ((p->n[1])*(p->n[2]))   )+
bound*(            +  ((p->n[1])*(p->n[2]))      )+   (  (iy+iz*(p->n[1]))    ));
  #else
    return (   4*field*(p->n[1]) +bound*((p->n[1]))  +   (iy)   );
  #endif
}


__device__ __host__
int encodempiw1 (struct params *p,int ix, int iy, int iz, int field,int bound) {
  #ifdef USE_SAC_3D
    return (4*field*(         ((p->n[0])*(p->n[2]))   )+
bound*(            +  ((p->n[0])*(p->n[2]))      )+   (  (ix+iz*(p->n[0]))    ));
  #else
    return (4*field*(p->n[0]) +bound*((p->n[0]))  +   (ix));
  #endif
}

__device__ __host__
int encodempiw2 (struct params *p,int ix, int iy, int iz, int field,int bound) {
  #ifdef USE_SAC_3D
    return (4*field*(         ((p->n[0])*(p->n[1]))   )+
bound*(            +  ((p->n[0])*(p->n[1]))      )+   (  (ix+iy*(p->n[0]))    ));
  #endif
}


__device__ __host__
int encodempivisc (struct params *p,int ix, int iy, int iz, int bound,int dim) {
  #ifdef USE_SAC_3D
    return (dim*(    2*(         (((p->n[0])+2)*((p->n[1])+2))+(((p->n[1])+2)*((p->n[2])+2))+(((p->n[0])+2)*((p->n[2])+2))   )           )+
bound*(         (dim==2)*(((p->n[0])+2)*((p->n[1])+2))   +  (dim==0)*(((p->n[1])+2)*((p->n[2])+2))  +   (dim==1)*(((p->n[0])+2)*((p->n[2])+2))    )+   (  (ix+iz*((p->n[0])+2))*(dim==1)+(iy+iz*((p->n[1])+2))*(dim==0)+(iz+ix*((p->n[2])+2))*(dim==2)    ));
  #else
    return (   dim*(2*(  ((p->n[0])+2)+((p->n[1])+2)   ))      +bound*(    (dim==1)*((p->n[0])+2)+(dim==0)*((p->n[1])+2)  )  +   (ix*(dim==1)+iy*(dim==0))     );
  #endif
}


__device__ __host__
int encodempivisc0 (struct params *p,int ix, int iy, int iz, int bound,int dim) {
  #ifdef USE_SAC_3D
    return (
bound*(           (((p->n[1])+2)*((p->n[2])+2))      )+   (  (iy+iz*((p->n[1])+2))    ));
  #else
    return (   bound*(    ((p->n[1])+2)  )  +   iy     );
  #endif
}


__device__ __host__
int encodempivisc1 (struct params *p,int ix, int iy, int iz, int bound,int dim) {
  #ifdef USE_SAC_3D
    return (
bound*(           (((p->n[0])+2)*((p->n[2])+2))      )+   (  (ix+iz*((p->n[0])+2))    ));
  #else
    return (   bound*(    ((p->n[0])+2)  )  +   ix     );
  #endif
}

__device__ __host__
int encodempivisc2 (struct params *p,int ix, int iy, int iz, int bound,int dim) {
  #ifdef USE_SAC_3D
    return (
bound*(           (((p->n[0])+2)*((p->n[1])+2))      )+   (  (ix+iy*((p->n[0])+2))    ));
  #endif
}


#ifdef USE_MPI

     __device__ __host__ void mpiwtogpu(struct params *p,real *d_w,real *d_wmod,real *d_mpiw0,real *d_mpiwmod0,real *d_mpiw1,real *d_mpiwmod1,real *d_mpiw2,real *d_mpiwmod2,int *ii, int var, int dim)
    {

             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;




                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_w[encode3_mpiu(p,i,j,k,var)]=d_mpiw0[encodempiw0(p,i,j,k,var,bound)];
                   // d_wmod[encode3_mpiu(p,i,j,k,var)]=d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)];
                    // if(var==4 && ((p)->ipe)==0)                        
                    //    printf(" %d %d %d %d actual %d  mpi data%d %g\n",i,j,bound,dim,var,encodempiw0(p,i,j,k,var,bound),d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]);   


 

     
      
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    d_w[encode3_mpiu(p,i,j,k,var)]=d_mpiw0[encodempiw0(p,i,j,k,var,bound)];
                  //  d_wmod[encode3_mpiu(p,i,j,k,var)]=d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)];    


                          // if(p->ipe==0    && var==rho && dim==0 )
                          //   {
                             // for(int bound=0;bound<=1;bound++)
                             //   printf("mpiw0 %d %d %d %d %lg \n",dim,bound,i,j,d_mpiw0[encodempiw0(p,i,j,k,var,bound)]);
			     	//printf("mpiwmod0 %d %d %d %d %lg %lg\n\n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)],d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]);
                           //  }
          
                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_w[encode3_mpiu(p,i,j,k,var)]=d_mpiw1[encodempiw1(p,i,j,k,var,bound)];
                   // d_wmod[encode3_mpiu(p,i,j,k,var)]=d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)];              
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                   bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                    d_w[encode3_mpiu(p,i,j,k,var)]=d_mpiw1[encodempiw1(p,i,j,k,var,bound)];
                  //  d_wmod[encode3_mpiu(p,i,j,k,var)]=d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)];              
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_w[encode3_mpiu(p,i,j,k,var)]=d_mpiw2[encodempiw2(p,i,j,k,var,bound)];
                  //  d_wmod[encode3_mpiu(p,i,j,k,var)]=d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=2*(k==((p->n[2])-1))+(p->n[2])-k;
                    d_w[encode3_mpiu(p,i,j,k,var)]=d_mpiw2[encodempiw2(p,i,j,k,var,bound)];
                  //  d_wmod[encode3_mpiu(p,i,j,k,var)]=d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)];              
                }

     #endif
 
// if( (p->ipe)==0  && ((p)->it)==0 && (isnan(d_wmod[fencode3_mpiu(p,ii,rho)]) || d_wmod[fencode3_mpiu(p,ii,rho)]==0      ))
//        { 
//    	printf("nant %d %d %d %lg\n",ii[0],ii[1], dim, d_wmod[fencode3_mpiu(p,ii,rho)] );
//}

    }

     __device__ __host__ void mpiwmodtogpu(struct params *p,real *d_w,real *d_wmod,real *d_mpiw0,real *d_mpiwmod0,real *d_mpiw1,real *d_mpiwmod1,real *d_mpiw2,real *d_mpiwmod2,int *ii, int var, int dim, int order)
    {

             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;

 #ifdef USE_SAC_3D
	int ntot=((p->n[0]))*((p->n[1]))*((p->n[2]))*NVAR;
 #else
	int ntot=((p->n[0]))*((p->n[1]))*NVAR;
 #endif
                //remember only update the boundaries if they are mpiupper boundaries 
                //or an mpi period 


                if((i==0 || i==1) && dim==0 /* && ( ((p->mpilowerb[dim])==1) || ((p->boundtype[0][dim][0])==2))  */  )
                {              
                    bound=i;
                    d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)]=d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)];
                     //if(var==0 && ((p)->ipe)==0)                        
                     //   printf(" %d %d %d %d actual mpi data %d %g\n",i,j,bound,dim,encodempiw0(p,i,j,k,var,bound),d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]); 

       
      
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0 /* && ( ((p->mpiupperb[dim])==1) || ((p->boundtype[0][dim][0])==2)) */)               
                {
                    bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)]=d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)];  

                             //if(p->ipe==0    && var==rho && dim==0 )
                             //{
                             // for(int bound=0;bound<=1;bound++)
                             //   printf("mpiw0 %d %d %d %d %lg \n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]);
			     //	printf("mpiwmod0 %d %d %d %d %lg %lg\n\n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)],d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]);
                             //}
          
                }

              

                if((j==0 || j==1) && dim==1  /*&& ( ((p->mpilowerb[dim])==1) || ((p->boundtype[0][0][0])==2))*/   )              
                {              
                    bound=j;
                    d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)]=d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]; 
//if(i>=0 && i<10  &&  var==rhob)
//printf("nani %d %d %d %d  %lg %lg %d  \n",p->ipe,order,i,j, d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)],d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)],bound );

             
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1  /* && ( ((p->mpiupperb[dim])==1) || ((p->boundtype[0][0][0])==2)) */   )               
                {
                   bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                    d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)]=d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]; 
//if(i>=0 && i<10  &&  var==rhob)
//printf("nani %d %d %d %d  %lg %lg %d  \n",p->ipe,order,i,j, d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)],d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)],bound );

  

             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2  /* && ( ((p->mpilowerb[dim])==1) || ((p->boundtype[0][dim][0])==2))  */ )              
                {              
                    bound=k;
                    d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)]=d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2  /* && ( ((p->mpiupperb[dim])==1) || ((p->boundtype[0][dim][0])==2))  */ )               
                {
                    bound=2*(k==((p->n[2])-1))+(p->n[2])-k;
                    d_wmod[order*ntot+encode3_mpiu(p,i,j,k,var)]=d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)];              
                }

     #endif
 
// if( (p->ipe)==0  && ((p)->it)==0 && (isnan(d_wmod[fencode3_mpiu(p,ii,rho)]) || d_wmod[fencode3_mpiu(p,ii,rho)]==0      ))
//        { 
//    	printf("nant %d %d %d %lg\n",ii[0],ii[1], dim, d_wmod[fencode3_mpiu(p,ii,rho)] );
//}

    }



     __device__ __host__ void mpiwdtogpu(struct params *p,real *d_wd,real *d_mpiw0,real *d_mpiw1,real *d_mpiw2,int *ii, int var, int dim)
    {

             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;

                //remember only update the boundaries if they are mpiupper boundaries 
                //or an mpi period 



 
                if((i==0 || i==1) && dim==0  /* &&  ((p->mpilowerb[dim])==1)*/)
                {              
                    bound=i;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw0[encodempiw0(p,i,j,k,var,bound)];
                    // if(var==4 && ((p)->ipe)==0)                        
                    //    printf(" %d %d %d %d actual %d  mpi data%d %g\n",i,j,bound,dim,var,encodempiw0(p,i,j,k,var,bound),d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]);        
      
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0 /* &&  ((p->mpiupperb[dim])==1)*/)               
                {
                    bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw0[encodempiw0(p,i,j,k,var,bound)];
                }

              

                if((j==0 || j==1) && dim==1  /* &&  ((p->mpilowerb[dim])==1)*/)              
                {              
                    bound=j;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw1[encodempiw1(p,i,j,k,var,bound)];
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1 /* &&  ((p->mpiupperb[dim])==1)*/)               
                {
                   bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw1[encodempiw1(p,i,j,k,var,bound)];
                }


               /* if((i==0 || i==1) && (j==0 || j==1))
                {              
                    bound=i;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw0[encodempiw0(p,i,j,k,var,bound)];
                    // if(var==4 && ((p)->ipe)==0)                        
                    //    printf(" %d %d %d %d actual %d  mpi data%d %g\n",i,j,bound,dim,var,encodempiw0(p,i,j,k,var,bound),d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]);        
      
                }
                if((( j>=((p->n[1])-2)   ))  && (( i>=((p->n[0])-2)   )))               
                {
                   bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw1[encodempiw0(p,i,j,k,var,bound)];
                }*/
                



       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw2[encodempiw2(p,i,j,k,var,bound)];
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=2*(k==((p->n[2])-1))+(p->n[2])-k;
                    d_wd[encode3_mpiu(p,i,j,k,var)]=d_mpiw2[encodempiw2(p,i,j,k,var,bound)];
                }

     #endif
 


    }




__device__ __host__ void   mpivisctogpu(struct params *p,real *d_wtemp2,real *d_gmpivisc0,real *d_gmpivisc1,real *d_gmpivisc2,int *ii,  int dim)
{
                                
               int i,j,k,bound,var;
              var=0;
i=ii[0];
j=ii[1];
k=0;
                //remember only update the boundaries if they are mpiupper boundaries 
                //or an mpi period 
 
                if((i==0 ) && dim==0 /* && ( ((p->mpilowerb[dim])==1) || ((p->boundtype[0][dim][0])==2))*/)
                {              
                    bound=i;
                    d_wtemp2[encode3p2_mpiu(p,i,j,k,var)]=d_gmpivisc0[encodempivisc0(p,i,j,k,bound,dim)];
                    
                }
                else if((( i==((p->n[0])+1)   ))  && dim==0  /* && ( ((p->mpiupperb[dim])==1) || ((p->boundtype[0][dim][0])==2)) */ )               
                {
                    bound=1;
                    d_wtemp2[encode3p2_mpiu(p,i,j,k,var)]=d_gmpivisc0[encodempivisc0(p,i,j,k,bound,dim)];
                }

              

                if((j==0) && dim==1 /* && ( ((p->mpilowerb[dim])==1) || ((p->boundtype[0][dim][0])==2))*/)              
                {              
                    bound=j;
                    d_wtemp2[encode3p2_mpiu(p,i,j,k,var)]=d_gmpivisc1[encodempivisc1(p,i,j,k,bound,dim)];
                }            
                 else if((( j==((p->n[1])+1)   ))  && dim==1   /*&& ( ((p->mpiupperb[dim])==1) || ((p->boundtype[0][dim][0])==2))*/)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_mpiu(p,i,j,k,var)]=d_gmpivisc1[encodempivisc1(p,i,j,k,bound,dim)];
             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 ) && dim==2  /*&& ( ((p->mpilowerb[dim])==1) || ((p->boundtype[0][dim][0])==2))*/)              
                {              
                    bound=k;
                    d_wtemp2[encode3p2_mpiu(p,i,j,k,var)]=d_gmpivisc2[encodempivisc2(p,i,j,k,bound,dim)];
                }        
                 else if(((k==((p->n[2])+1)   ))  && dim==2   /* && ( ((p->mpiupperb[dim])==1) || ((p->boundtype[0][dim][0])==2))*/)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_mpiu(p,i,j,k,var)]=d_gmpivisc2[encodempivisc2(p,i,j,k,bound,dim)];
                }

     #endif
                               
                                
}

__device__ __host__ void   gputompivisc(struct params *p,real *d_wtemp2,real *d_gmpivisc0,real *d_gmpivisc1,real *d_gmpivisc2,int *ii,  int dim)
{
                                
              int i,j,k,bound,var;
              var=0;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 ) && dim==0)
                {              
                    bound=i;
                    d_gmpivisc0[encodempivisc0(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_mpiu(p,i,j,k,var)];
                    
                }
                else if((( i==((p->n[0])+1)   ))  && dim==0)               
                {
                    bound=1;
                    d_gmpivisc0[encodempivisc0(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_mpiu(p,i,j,k,var)];
                }

              

                if((j==0) && dim==1)              
                {              
                    bound=j;
                    d_gmpivisc1[encodempivisc1(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_mpiu(p,i,j,k,var)];
                }            
                 else if((( j==((p->n[1])+1)   ))  && dim==1)               
                {
                    bound=1;
                    d_gmpivisc1[encodempivisc1(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_mpiu(p,i,j,k,var)];
             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 ) && dim==2)              
                {              
                    bound=k;
                    d_gmpivisc2[encodempivisc2(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_mpiu(p,i,j,k,var)];
                }        
                 else if(((k==((p->n[2])+1)   ))  && dim==2)               
                {
                    bound=1;
                    d_gmpivisc2[encodempivisc2(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_mpiu(p,i,j,k,var)];
                }

     #endif
                               
                                
}

     __device__ __host__ void gputompiw(struct params *p,real *d_w,real *d_wmod,real *d_mpiw0,real *d_mpiwmod0,real *d_mpiw1,real *d_mpiwmod1,real *d_mpiw2,real *d_mpiwmod2,int *ii, int var, int dim)
    {
             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;

/*  for(int field=rho;field<=rho ; field++)
if(  (p->ipe)==0  && ((p)->it)==1 && ( isnan(d_wmod[fencode3_mpiu(p,ii,field)])|| d_wmod[fencode3_mpiu(p,ii,field)]==0 ))
        { 
    				printf("nant %d %d %d %d %lg %lg \n",ii[0],ii[1],field,dim,d_wmod[fencode3_mpiu(p,ii,rho)],d_wmod[fencode3_mpiu(p,ii,field)] );
}*/
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_mpiw0[encodempiw0(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i+2,j,k,var)];
                   // d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i+2,j,k,var)];



              
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    d_mpiw0[encodempiw0(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i-2,j,k,var)];
                   // d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i-2,j,k,var)];  




                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_mpiw1[encodempiw1(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j+2,k,var)];
                   // d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i,j+2,k,var)];   








			/*  if( var==0 && (p)->ipe==3 && ((p)->it)==2 && bound==3)
			    {
				 //printf("ipe3 mpiwmod \n");
				 //for(int iii=0; iii<4*((p)->n[0]);iii++)
				     printf(" %lg %d \n",d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)],encodempiw1(p,i,j,k,var,bound));
				 //printf("\n");
			     }*/


           
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                    d_mpiw1[encodempiw1(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j-2,k,var)];
                   // d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i,j-2,k,var)];  

                          /*   if(p->ipe==0    && var==rho && dim==1 )
                             {
                             // for(int bound=0;bound<=1;bound++)
                                printf("mpiw0 %d %d %d %d %lg \n",dim,bound,i,j,d_mpiw0[encodempiw0(p,i,j,k,var,bound)]);
			     	//printf("mpiwmod0 %d %d %d %d %lg %lg\n\n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)],d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]);
                             }*/



             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_mpiw2[encodempiw2(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j,k+2,var)];
                   // d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i,j,k+2,var)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=2*(k==((p->n[2])-1))+(p->n[2])-k;
                    d_mpiw2[encodempiw2(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j,k-2,var)];
                   // d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i,j,k-2,var)];               
                }

     #endif



    /*if((p)->ipe==3 && ((p)->it)==2  && i==((p->n[0])-1) && j==((p->n[1])-1))
    {
         printf("ipe3 mpiwmod \n");
         for(int iii=0; iii<4*((p)->n[0]);iii++)
             printf(" %lg ",d_mpiwmod1[iii]);
         printf("\n");
     }*/

 
 
 }


     __device__ __host__ void gputompiwmod(struct params *p,real *d_w,real *d_wmod,real *d_mpiw0,real *d_mpiwmod0,real *d_mpiw1,real *d_mpiwmod1,real *d_mpiw2,real *d_mpiwmod2,int *ii, int var, int dim, int order)
    {
             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;

 #ifdef USE_SAC_3D
	int ntot=((p->n[0]))*((p->n[1]))*((p->n[2]))*NVAR;
 #else
	int ntot=((p->n[0]))*((p->n[1]))*NVAR;
 #endif

/*  for(int field=rho;field<=rho ; field++)
if(  (p->ipe)==0  && ((p)->it)==1 && ( isnan(d_wmod[fencode3_mpiu(p,ii,field)])|| d_wmod[fencode3_mpiu(p,ii,field)]==0 ))
        { 
    				printf("nant %d %d %d %d %lg %lg \n",ii[0],ii[1],field,dim,d_wmod[fencode3_mpiu(p,ii,rho)],d_wmod[fencode3_mpiu(p,ii,field)] );
}*/
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]=d_wmod[(order*ntot)+encode3_mpiu(p,i+2,j,k,var)];
                   // d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i+2,j,k,var)];

                            // if(p->ipe==0    && var==rho && dim==0 )
                             //{
                             // for(int bound=0;bound<=1;bound++)
                              //  printf("mpiw0 %d %d %d %d %lg \n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]);
			     	//printf("mpiwmod0 %d %d %d %d %lg %lg\n\n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)],d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]);
                             //}
              
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    //d_mpiw0[encodempiw0(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i-2,j,k,var)];
                    d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]=d_wmod[(order*ntot)+encode3_mpiu(p,i-2,j,k,var)];  

                             //if(p->ipe==0    && var==rho && dim==0 )
                             //{
                             // for(int bound=0;bound<=1;bound++)
                             //   printf("mpiw0 %d %d %d %d %lg \n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]);
			     	//printf("mpiwmod0 %d %d %d %d %lg %lg\n\n",dim,bound,i,j,d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)],d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]);
                             //}





                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    //d_mpiw1[encodempiw1(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j+2,k,var)];
                    d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]=d_wmod[(order*ntot)+encode3_mpiu(p,i,j+2,k,var)];   






			/*  if( var==0 && (p)->ipe==3 && ((p)->it)==2 && bound==3)
			    {
				 //printf("ipe3 mpiwmod \n");
				 //for(int iii=0; iii<4*((p)->n[0]);iii++)
				     printf(" %lg %d \n",d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)],encodempiw1(p,i,j,k,var,bound));
				 //printf("\n");
			     }*/


           
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                   // d_mpiw1[encodempiw1(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j-2,k,var)];
                    d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)]=d_wmod[(order*ntot)+encode3_mpiu(p,i,j-2,k,var)];  





             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                   // d_mpiw2[encodempiw2(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j,k+2,var)];
                    d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)]=d_wmod[(order*ntot)+encode3_mpiu(p,i,j,k+2,var)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=2*(k==((p->n[2])-1))+(p->n[2])-k;
                    //d_mpiw2[encodempiw2(p,i,j,k,var,bound)]=d_w[encode3_mpiu(p,i,j,k-2,var)];
                    d_mpiwmod2[encodempiw2(p,i,j,k,var,bound)]=d_wmod[(order*ntot)+encode3_mpiu(p,i,j,k-2,var)];               
                }

     #endif



    /*if((p)->ipe==3 && ((p)->it)==2  && i==((p->n[0])-1) && j==((p->n[1])-1))
    {
         printf("ipe3 mpiwmod \n");
         for(int iii=0; iii<4*((p)->n[0]);iii++)
             printf(" %lg ",d_mpiwmod1[iii]);
         printf("\n");
     }*/

 
 
 }





     __device__ __host__ void gputompiwd(struct params *p,real *d_wd,real *d_mpiw0,real *d_mpiw1,real *d_mpiw2,int *ii, int var, int dim)
    {
             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_mpiw0[encodempiw0(p,i,j,k,var,bound)]=d_wd[encode3_mpiu(p,i+2,j,k,var)];
                 }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    d_mpiw0[encodempiw0(p,i,j,k,var,bound)]=d_wd[encode3_mpiu(p,i-2,j,k,var)];
                  }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_mpiw1[encodempiw1(p,i,j,k,var,bound)]=d_wd[encode3_mpiu(p,i,j+2,k,var)];
 

			/*  if( var==0 && (p)->ipe==3 && ((p)->it)==2 && bound==3)
			    {
				 //printf("ipe3 mpiwmod \n");
				 //for(int iii=0; iii<4*((p)->n[0]);iii++)
				     printf(" %lg %d \n",d_mpiwmod1[encodempiw1(p,i,j,k,var,bound)],encodempiw1(p,i,j,k,var,bound));
				 //printf("\n");
			     }*/


           
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=2*(j==((p->n[1])-1))+(p->n[1])-j;
                    d_mpiw1[encodempiw1(p,i,j,k,var,bound)]=d_wd[encode3_mpiu(p,i,j-2,k,var)];
                 }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_mpiw2[encodempiw2(p,i,j,k,var,bound)]=d_wd[encode3_mpiu(p,i,j,k+2,var)];
                 }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=2*(k==((p->n[2])-1))+(p->n[2])-k;
                    d_mpiw2[encodempiw2(p,i,j,k,var,bound)]=d_wd[encode3_mpiu(p,i,j,k-2,var)];
                 }

     #endif



    /*if((p)->ipe==3 && ((p)->it)==2  && i==((p->n[0])-1) && j==((p->n[1])-1))
    {
         printf("ipe3 mpiwmod \n");
         for(int iii=0; iii<4*((p)->n[0]);iii++)
             printf(" %lg ",d_mpiwmod1[iii]);
         printf("\n");
     }*/

 
 
 }






__global__ void  mpiwtogpu_parallel(struct params *p,real *d_w, real *d_wmod, real *d_mpiw0, real *d_mpiwmod0, real *d_mpiw1, real *d_mpiwmod1, real *d_mpiw2, real *d_mpiwmod2, int idir)
{

int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

#ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;

//if(iindex==0)
//         printf("in mpiwtogpu\n");

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      //for(int dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b3; f++)
            for( f=rho; f<NVAR; f++)
     #else
     //for(int dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b2; f++)
	  for( f=rho; f<NVAR; f++)
     #endif     
         #ifdef USE_SAC_3D
           if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
         #else
           if(i<((p->n[0])) && j<((p->n[1])))
         #endif     
{      
    // if(iindex==0)
    //     printf("calling  mpiwtogpu %d %d\n",dim,f);

                 mpiwtogpu(p,d_w,d_wmod,d_mpiw0,d_mpiwmod0,d_mpiw1,d_mpiwmod1,d_mpiw2,d_mpiwmod2,iia,f,idir);

}


 __syncthreads();

           
               
}



__global__ void  mpiwmodtogpu_parallel(struct params *p,real *d_w, real *d_wmod, real *d_mpiw0, real *d_mpiwmod0, real *d_mpiw1, real *d_mpiwmod1, real *d_mpiw2, real *d_mpiwmod2, int idir, int order)
{

int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

//int var,bound;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

#ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;

//if(iindex==0)
//         printf("in mpiwtogpu\n");

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      //for(int dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b3; f++)
            for( f=rho; f<NVAR; f++)
     #else
     //for(int dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b2; f++)
	  for( f=rho; f<NVAR; f++)
     #endif     
         #ifdef USE_SAC_3D
           if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
         #else
           if(i<((p->n[0])) && j<((p->n[1])))
         #endif     
{      
    // if(iindex==0)
    //     printf("calling  mpiwtogpu %d %d\n",dim,f);

                    //bound=2*(i==((p->n[0])-1))+(p->n[0])-i;
                    /*bound=i;
                    var=f;
                    d_mpiwmod0[encodempiw0(p,i,j,k,var,bound)]=d_wmod[encode3_mpiu(p,i+2,j,k,var)];*/


                 //if( f==rho && (p->ipe)==3 && (iia[0]==0  || iia[0]==1) )
                      // if(idir==0)
		      // {
		       //d_mpiwmod0[encodempiw0(p,iia[0],iia[1],iia[2],f,iia[0])]=j+(p->ipe)*(1000);
    		 		//printf("nani0 %d %d %d  %lg  \n",p->ipe, iia[0],iia[1], d_mpiwmod0[encodempiw0(p,iia[0],iia[1],iia[2],f,iia[0])]); 
				//}


 mpiwmodtogpu(p,d_w,d_wmod,d_mpiw0,d_mpiwmod0,d_mpiw1,d_mpiwmod1,d_mpiw2,d_mpiwmod2,iia,f,idir,order);

                
		/*int bound;
                 if( f==rhob /*&& (p->ipe)==1  && (  d_wmod[fencode3_mpiu(p,iia,f)]==0 )*/ /*  && (j==0 || j==513)   && i>=0 && i<20)
                       if(idir==1)
                       {

			if(j==0 || j==1)
				bound=j;

                           if(j>=((p->n[1])-2))
 				bound=2*(j==((p->n[1])-1))+(p->n[1])-j;


    				printf("nani %d %d %d  %lg %lg %d  \n",p->ipe,iia[0],iia[1], d_mpiwmod1[encodempiw1(p,i,j,k,f,bound)],d_wmod[encode3_mpiu(p,i+2,j,k,f)],bound );
				
                       }*/


                 //if( f==rho && (p->ipe)==3 && (iia[0]==0  || iia[0]==1) )
                       //if(idir==0)
    		 		//printf("nani0 %d %d  %lg %lg \n",iia[0],iia[1], d_wmod[fencode3_mpiu(p,iia,rho)],d_wmod[fencode3_mpiu(p,iia,f)+dimp*NVAR] );

}


 __syncthreads();

           
               
}




__global__ void  mpiwdtogpu_parallel(struct params *p,  int dir, int var, real *d_wd, real *d_mpiw0,  real *d_mpiw1,  real *d_mpiw2, int idir)
{

int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

#ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;

//if(iindex==0)
//         printf("in mpiwtogpu\n");

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      //for(int dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b3; f++)
            for( f=pos1; f<=delx3; f++)
     #else
     //for(int dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b2; f++)
	  for( f=pos1; f<=delx2; f++)
     #endif     
         #ifdef USE_SAC_3D
           if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
         #else
           if(i<((p->n[0])) && j<((p->n[1])))
         #endif     
{      
    // if(iindex==0)
    //     printf("calling  mpiwtogpu %d %d\n",dim,f);

                 mpiwdtogpu(p,d_wd,d_mpiw0,d_mpiw1,d_mpiw2,iia,f,idir);

}


 __syncthreads();

           
               
}



     __global__ void gputompiwmod_parallel(struct params *p,real *d_w,real *d_wmod,real *d_mpiw0,real *d_mpiwmod0,real *d_mpiw1,real *d_mpiwmod1,real *d_mpiw2,real *d_mpiwmod2,int order, int idir)
    {

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      //for(dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b3; f++)
           for( f=rho; f<NVAR; f++)
     #else
           //for(dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b2; f++)
	   for( f=rho; f<NVAR; f++)
     #endif
             {
            
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{

 

                  gputompiwmod(p,d_w,d_wmod,d_mpiw0,d_mpiwmod0,d_mpiw1,d_mpiwmod1,d_mpiw2,d_mpiwmod2,iia,f,idir,order);

                             /*if( f==rho && idir==0 )
                                          if((i==0 || i==1) )
						{
                                 		
							d_mpiwmod0[encodempiw0(p,i,j,k,f,i)]=1000*(p->ipe)+j;
							//printf("mpiw0 %d %d %d %d %d %lg %lg\n",p->ipe,idir,i,iia[0],iia[1],d_mpiwmod0[encodempiw0(p,i,j,k,f,i)],d_mpiwmod1[encodempiw1(p,i,j,k,f,i)]);
                                                  }*/


                            // if(p->ipe==3    && f==rho && idir==0 )
                                       //   if((i==0 || i==1) )
						//{
                                 		
							//d_mpiwmod0[encodempiw0(p,i,j,k,f,i)]=1000*(p->ipe)+j;
							//printf("mpiw0 %d %d %d %d %d %lg %lg\n",p->ipe,idir,i,iia[0],iia[1],d_mpiwmod0[encodempiw0(p,i,j,k,f,i)],d_mpiwmod1[encodempiw1(p,i,j,k,f,i)]);
                                                 // }

	}

               }





 __syncthreads();

}

     __global__ void gputompiw_parallel(struct params *p,real *d_w,real *d_wmod,real *d_mpiw0,real *d_mpiwmod0,real *d_mpiw1,real *d_mpiwmod1,real *d_mpiw2,real *d_mpiwmod2,int order, int idir)
    {

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      //for(dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b3; f++)
           for( f=rho; f<NVAR; f++)
     #else
           //for(dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b2; f++)
	   for( f=rho; f<NVAR; f++)
     #endif
             {
            
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{

 

                  gputompiw(p,d_w,d_wmod,d_mpiw0,d_mpiwmod0,d_mpiw1,d_mpiwmod1,d_mpiw2,d_mpiwmod2,iia,f,idir);

                             //if(p->ipe==0    && f==rho && idir==0 )
                             // for(int bound=0;bound<=1;bound++)
                             //   printf("mpiw0 %d %d %d %d %lg %lg\n",idir,bound,iia[0],iia[1],d_mpiw0[encodempiw0(p,i,j,k,f,bound)],d_mpiw1[encodempiw1(p,i,j,k,f,bound)]);

	}

               }





 __syncthreads();

}


     __global__ void gputompiwd_parallel(struct params *p,real *d_wd,real *d_mpiw0,real *d_mpiw1,real *d_mpiw2,int order, int idir)
    {

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      //for(dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b3; f++)
           for( f=pos1; f<=delx3; f++)
     #else
        //   for(dim=0; dim<NDIM;dim++)
           //for( f=rho; f<=b2; f++)
	   for( f=pos1; f<=delx2; f++)
     #endif
             {
            
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{

 

                  gputompiwd(p,d_wd,d_mpiw0,d_mpiw1,d_mpiw2,iia,f,idir);


	}

               }





 __syncthreads();

}


     __global__ void gputompivisc_parallel(struct params *p,real *d_wtemp2,real *d_gmpivisc0,real *d_gmpivisc1,real *d_gmpivisc2)
     {
               
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/((nj+2)*(ni+2));
   jp=(iindex-(kp*((nj+2)*(ni+2))))/(ni+2);
   ip=iindex-(kp*(nj+2)*(ni+2))-(jp*(ni+2));
#else
    jp=iindex/(ni+2);
   ip=iindex-(jp*(ni+2));
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];

     #else

     #endif
           for(dim=0; dim<NDIM;dim++)
             {
            
         #ifdef USE_SAC_3D
      if(i<(((p->n[0])+2)) && j<(((p->n[1])+2))  && k<(((p->n[2])+2)))
     #else
       if(i<(((p->n[0])+2)) && j<(((p->n[1])+2)))
     #endif           
	{

 

                  gputompivisc(p,d_wtemp2,d_gmpivisc0,d_gmpivisc1,d_gmpivisc2,iia,dim);

	}

               }

 __syncthreads();
              
               }    
     
     
    __global__ void  mpivisctogpu_parallel(struct params *p,real *d_wtemp2,real *d_gmpivisc0,real *d_gmpivisc1,real *d_gmpivisc2)
    {
               
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/((nj+2)*(ni+2));
   jp=(iindex-(kp*((nj+2)*(ni+2))))/(ni+2);
   ip=iindex-(kp*(nj+2)*(ni+2))-(jp*(ni+2));
#else
    jp=iindex/(ni+2);
   ip=iindex-(jp*(ni+2));
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];

     #else

     #endif
           for(dim=0; dim<NDIM;dim++)
             {
            
         #ifdef USE_SAC_3D
      if(i<(((p->n[0])+2)) && j<(((p->n[1])+2))  && k<(((p->n[2])+2)))
     #else
       if(i<(((p->n[0])+2)) && j<(((p->n[1])+2)))
     #endif           
	{

 

                  mpivisctogpu(p,d_wtemp2,d_gmpivisc0,d_gmpivisc1,d_gmpivisc2,iia,dim);

	}

               }

 __syncthreads();
               
               
}

#endif



/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_mpiu(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
  
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  


}



#ifdef USE_MULTIGPU




//prepare data buffers used to copy data between gpu and cpu
//this will update only the ghost cells transferred between the CPU's

int cuinitmgpurbuffers(struct params **p,    
real **d_gmpiviscr0,    
real **d_gmpiviscr1,    
real **d_gmpiviscr2,   
real **d_gmpiwr0, 
real **d_gmpiwmodr0,   
real **d_gmpiwr1, 
real **d_gmpiwmodr1,   
real **d_gmpiwr2, 
real **d_gmpiwmodr2)
{

  int szw,  szvisc0,szvisc1,szvisc2,szw0,szw1,szw2;
  #ifdef USE_SAC
  //real *dt;
  
  szw=4*(  ((*p)->n[1])  +  ((*p)->n[0])   );
  szw0=4*NDERV*(  ((*p)->n[1])     );
  szw1=4*NDERV*(  ((*p)->n[0])     );

  szvisc0=4*(  (((*p)->n[1])+2 )   );
  szvisc1=4*(    (((*p)->n[0]) +2 )  );

 //dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2),sizeof(real));

  #endif
  #ifdef USE_SAC_3D
  
  szw=4*NDERV*(  ((*p)->n[1])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[1])  );
  szw0=4*NDERV*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NDERV*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NDERV*(    ((*p)->n[0])*((*p)->n[1])  );



  szvisc0=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)  ); 
  szvisc1=4*(   (((*p)->n[0])+2)*(((*p)->n[2])+2)    );    
  szvisc2=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)   );    

   
  //dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2),sizeof(real));
  #endif

  	  hipMalloc((void**)d_gmpiwmodr0, szw0*sizeof(real));
	  hipMalloc((void**)d_gmpiwr0, szw0*sizeof(real));
	  hipMalloc((void**)d_gmpiwmodr1, szw1*sizeof(real));
	  hipMalloc((void**)d_gmpiwr1, szw1*sizeof(real));

  #ifdef USE_SAC_3D  
	  hipMalloc((void**)d_gmpiwmodr2, szw2*sizeof(real));
	  hipMalloc((void**)d_gmpiwr2, szw2*sizeof(real));
          hipMalloc((void**)d_gmpiviscr2, szvisc2*sizeof(real));
  #else

          hipMalloc((void**)d_gmpiviscr2, sizeof(real));
  #endif
          hipMalloc((void**)d_gmpiviscr0, szvisc0*sizeof(real));
          hipMalloc((void**)d_gmpiviscr1, szvisc1*sizeof(real));
  return 0;



}





//prepare data buffers used to copy data between gpu and cpu
//this will update only the ghost cells transferred between the CPU's


int cuinitmgpubuffers(struct params **p,real **w, real **wmod, real **temp2, real **gmpivisc0, real **gmpivisc1, real **gmpivisc2,   real **gmpiw0, real **gmpiwmod0,   real **gmpiw1, real **gmpiwmod1,   real **gmpiw2, real **gmpiwmod2, struct params **d_p,   real **d_w, real **d_wmod,real **d_wtemp2,    real **d_gmpivisc0,    real **d_gmpivisc1,    real **d_gmpivisc2,   real **d_gmpiw0, real **d_gmpiwmod0,   real **d_gmpiw1, real **d_gmpiwmod1,   real **d_gmpiw2, real **d_gmpiwmod2)
{

  int szw,  szvisc0,szvisc1,szvisc2,szw0,szw1,szw2;
  #ifdef USE_SAC
  //real *dt;
  
  szw=4*(  ((*p)->n[1])  +  ((*p)->n[0])   );
  szw0=4*NDERV*(  ((*p)->n[1])     );
  szw1=4*NDERV*(  ((*p)->n[0])     );

  szvisc0=4*(  (((*p)->n[1])+2 )   );
  szvisc1=4*(    (((*p)->n[0]) +2 )  );

 //dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2),sizeof(real));

  #endif
  #ifdef USE_SAC_3D
  
  szw=4*NDERV*(  ((*p)->n[1])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[1])  );
  szw0=4*NDERV*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NDERV*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NDERV*(    ((*p)->n[0])*((*p)->n[1])  );



  szvisc0=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)  ); 
  szvisc1=4*(   (((*p)->n[0])+2)*(((*p)->n[2])+2)    );    
  szvisc2=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)   );    

   
  //dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2),sizeof(real));
  #endif






  //temp2=&dt;
  //gmpiwmod=(real **)malloc(szw*sizeof(real));
  //gmpiw=(real **)malloc(szw*sizeof(real));

  //gmpiwmod0=(real **)malloc(szw0*sizeof(real));
  //gmpiw0=(real **)malloc(szw0*sizeof(real));
  //gmpiwmod1=(real **)malloc(szw1*sizeof(real));
  //gmpiw1=(real **)malloc(szw1*sizeof(real));

  #ifdef USE_SAC_3D
	//  gmpiwmod2=(real **)malloc(szw2*sizeof(real));
	//  gmpiw2=(real **)malloc(szw2*sizeof(real));
  #endif

  //gmpivisc=(real **)malloc(szvisc*sizeof(real));
	//  hipMalloc((void**)d_gmpiwmod, szw*sizeof(real));
	//  hipMalloc((void**)d_gmpiw, szw*sizeof(real));


  	  hipMalloc((void**)d_gmpiwmod0, szw0*sizeof(real));
	  hipMalloc((void**)d_gmpiw0, szw0*sizeof(real));
	  hipMalloc((void**)d_gmpiwmod1, szw1*sizeof(real));
	  hipMalloc((void**)d_gmpiw1, szw1*sizeof(real));

  #ifdef USE_SAC_3D  
	  hipMalloc((void**)d_gmpiwmod2, szw2*sizeof(real));
	  hipMalloc((void**)d_gmpiw2, szw2*sizeof(real));
          hipMalloc((void**)d_gmpivisc2, szvisc2*sizeof(real));
  #else

          hipMalloc((void**)d_gmpivisc2, sizeof(real));
  #endif
          hipMalloc((void**)d_gmpivisc0, szvisc0*sizeof(real));
          hipMalloc((void**)d_gmpivisc1, szvisc1*sizeof(real));
  return 0;
}

//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopywtompiwmod(struct params **p,real **w, real **wmod,    real **gmpiw0, real **gmpiwmod0,    real **gmpiw1, real **gmpiwmod1,    real **gmpiw2, real **gmpiwmod2, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw0, real **d_gmpiwmod0,   real **d_gmpiw1, real **d_gmpiwmod1,   real **d_gmpiw2, real **d_gmpiwmod2, int order, int idir)
{
     int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;
     int szw0,szw1,szw2;

     int szbuf;
     int dimp=(((*p)->n[0]))*(((*p)->n[1]));
     
     
     i3=0;
     #ifdef USE_SAC_3D  
       dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
     #endif 
     int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif


  #ifdef USE_SAC
  
  szw0=4*NVAR*(  ((*p)->n[1])     );
  szw1=4*NVAR*(  ((*p)->n[0])     );

  #endif
  #ifdef USE_SAC_3D
  
   szw0=4*NVAR*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NVAR*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NVAR*(    ((*p)->n[0])*((*p)->n[1])  );

  #endif

    //real **d_tgmpiw0;
    //real **tgmpiw0=(real **)malloc(szw0*sizeof(real));
    //hipMalloc((void**)d_tgmpiw0, szw0*sizeof(real));
    // for(var=0; var<NVAR; var++)
    //   for(dim=0;dim<NDIM;dim++)
     gputompiwmod_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw0,*d_gmpiwmod0,*d_gmpiw1,*d_gmpiwmod1,*d_gmpiw2,*d_gmpiwmod2,order,idir);

 
 #ifdef USE_GPUDIRECT
    // printf("using gpudirect in1\n");
     hipDeviceSynchronize();

#else
 
     
     hipDeviceSynchronize();

if(idir==0)
{
     hipMemcpy(*gmpiwmod0, *d_gmpiwmod0, szw0*sizeof(real), hipMemcpyDeviceToHost);

// hipDeviceSynchronize();

   //   hipMemcpy(*gmpiw0, *d_gmpiw0, szw0*sizeof(real), hipMemcpyDeviceToHost);




}

if(idir==1)
{
     hipMemcpy(*gmpiwmod1, *d_gmpiwmod1, szw1*sizeof(real), hipMemcpyDeviceToHost);
   //  hipMemcpy(*gmpiw1, *d_gmpiw1, szw1*sizeof(real), hipMemcpyDeviceToHost);
}
 

//struct params *tp;     







    

   #ifdef USE_SAC_3D
if(idir==2)
{
     hipMemcpy(*gmpiwmod2, *d_gmpiwmod2, szw2*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*gmpiw2, *d_gmpiw2, szw2*sizeof(real), hipMemcpyDeviceToHost);
}
   #endif 
   
   
   #endif

hipDeviceSynchronize();
}


//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopywtompiw(struct params **p,real **w, real **wmod,    real **gmpiw0, real **gmpiwmod0,    real **gmpiw1, real **gmpiwmod1,    real **gmpiw2, real **gmpiwmod2, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw0, real **d_gmpiwmod0,   real **d_gmpiw1, real **d_gmpiwmod1,   real **d_gmpiw2, real **d_gmpiwmod2, int order, int idir)
{
     int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;
     int szw0,szw1,szw2;

     int szbuf;
     int dimp=(((*p)->n[0]))*(((*p)->n[1]));
     
     
     i3=0;
     #ifdef USE_SAC_3D  
       dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
     #endif 
     int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif


  #ifdef USE_SAC
  
  szw0=4*NVAR*(  ((*p)->n[1])     );
  szw1=4*NVAR*(  ((*p)->n[0])     );

  #endif
  #ifdef USE_SAC_3D
  
   szw0=4*NVAR*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NVAR*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NVAR*(    ((*p)->n[0])*((*p)->n[1])  );

  #endif

    //real **d_tgmpiw0;
    //real **tgmpiw0=(real **)malloc(szw0*sizeof(real));
    //hipMalloc((void**)d_tgmpiw0, szw0*sizeof(real));
    // for(var=0; var<NVAR; var++)
    //   for(dim=0;dim<NDIM;dim++)
     gputompiw_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw0,*d_gmpiwmod0,*d_gmpiw1,*d_gmpiwmod1,*d_gmpiw2,*d_gmpiwmod2,order,idir);


#ifdef USE_GPUDIRECT
     
     hipDeviceSynchronize();

#else



     
     hipDeviceSynchronize();

if(idir==0)
{
     //hipMemcpy(*gmpiwmod0, *d_gmpiwmod0, szw0*sizeof(real), hipMemcpyDeviceToHost);


      hipMemcpy(*gmpiw0, *d_gmpiw0, szw0*sizeof(real), hipMemcpyDeviceToHost);
}

if(idir==1)
{
     //hipMemcpy(*gmpiwmod1, *d_gmpiwmod1, szw1*sizeof(real), hipMemcpyDeviceToHost);
     hipMemcpy(*gmpiw1, *d_gmpiw1, szw1*sizeof(real), hipMemcpyDeviceToHost);
}
      
    

   #ifdef USE_SAC_3D
if(idir==2)
{
    // hipMemcpy(*gmpiwmod2, *d_gmpiwmod2, szw2*sizeof(real), hipMemcpyDeviceToHost);
     hipMemcpy(*gmpiw2, *d_gmpiw2, szw2*sizeof(real), hipMemcpyDeviceToHost);
}
   #endif 
   
   #endif

hipDeviceSynchronize();


 /*if(((*p)->ipe)==3  && ((*p)->it)==2)
{


       printf("%d %d \n",szw0,szw1);

        for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=0;
                       bound=0;
                       var=0;
                            printf(" %d %d %d %lg %d \n",i1,i2,bound,(*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)],encodempiw1(*p,i1,i2,i3,var,bound));                                        
                     ;//  (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)];              
                     ;//  (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)];



                  }
         ;// printf("\n");
}*/









//encodempiw1(p,i,j,k,var,bound)]



   //free(tgmpiw0);
   //hipFree(*d_tgmpiw0);
//gmpiw behaving OK but cannot display or access any of the gmpiwmod variables!
//printf("%f\n",(*gmpiwmod)[0]);
     
//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
   /*  for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC_3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)];              
                       (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)];
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                      


		
                       (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)];  
                      //if(encodempiw(*p,i1,i2,i3,var,bound,dim)<10)  
                      if(var==5 && ((*p)->ipe)==0)                        
                        printf(" %d %d %d %d actual %d  mpi data%d %g\n",i1,i2,bound,dim,var,encodempiw0(*p,i1,i2,i3,var,bound),(*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)]);

                     ;// if(encodempiw(*p,i1,i2,i3,var,bound,dim)<10239 )
                       (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)];
                                                                     
                      // *(wmod+encode3_mpiu(*p,ii,var))=*(gmpiwmod0+encodempiw0(*p,i1,i2,i3,var,bound));              
                      // (*w)[encode3_mpiu(*p,ii,var)]=(*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)];

                      /* if(var==4  && ((*p)->ipe)==1)
                       {
				(*wmod)[fencode3_mpiu(*p,ii,var)]=0.5;
				(*w)[fencode3_mpiu(*p,ii,var)]=0.5;
                       }*/


              /*    }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC_3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)];              
                       (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)];
                  }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                                                                     
                     ;//  (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)];              
                     ;//  (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)];



                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC_3D
                       case 2:
         i3=bound*(bound<2)+( ((*p)->n[2])-(bound-1) )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod2)[encodempiw2(*p,i1,i2,i3,var,bound)];              
                       (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw2)[encodempiw2(*p,i1,i2,i3,var,bound)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    */

}




int cucopywdtompiwd(struct params **p,real **wd,    real **gmpiw0,    real **gmpiw1,    real **gmpiw2, struct params **d_p  ,real **d_wd,   real **d_gmpiw0,   real **d_gmpiw1,   real **d_gmpiw2, int order, int idir)
{
     int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;
     int szw0,szw1,szw2;

     int szbuf;
     int dimp=(((*p)->n[0]))*(((*p)->n[1]));
     
     
     i3=0;
     #ifdef USE_SAC_3D  
       dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
     #endif 
     int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif


  #ifdef USE_SAC
  
  szw0=4*NDERV*(  ((*p)->n[1])     );
  szw1=4*NDERV*(  ((*p)->n[0])     );

  #endif
  #ifdef USE_SAC_3D
  
   szw0=4*NDERV*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NDERV*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NDERV*(    ((*p)->n[0])*((*p)->n[1])  );

  #endif

    //real **d_tgmpiw0;
    //real **tgmpiw0=(real **)malloc(szw0*sizeof(real));
    //hipMalloc((void**)d_tgmpiw0, szw0*sizeof(real));
    // for(var=0; var<NVAR; var++)
    //   for(dim=0;dim<NDIM;dim++)
     gputompiwd_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wd,*d_gmpiw0,*d_gmpiw1,*d_gmpiw2,order, idir);

;//#ifdef USE_GPUDIRECT
     
;//     hipDeviceSynchronize();

;//#else


     
     hipDeviceSynchronize();
if(idir==0)
      hipMemcpy(*gmpiw0, *d_gmpiw0, szw0*sizeof(real), hipMemcpyDeviceToHost);

if(idir==1)
     hipMemcpy(*gmpiw1, *d_gmpiw1, szw1*sizeof(real), hipMemcpyDeviceToHost);

      
    

   #ifdef USE_SAC_3D
if(idir==2)
      hipMemcpy(*gmpiw2, *d_gmpiw2, szw2*sizeof(real), hipMemcpyDeviceToHost);
   #endif 

hipDeviceSynchronize();
;//#endif

 /*if(((*p)->ipe)==3  && ((*p)->it)==2)
{


       printf("%d %d \n",szw0,szw1);

        for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=0;
                       bound=0;
                       var=0;
                            printf(" %d %d %d %lg %d \n",i1,i2,bound,(*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)],encodempiw1(*p,i1,i2,i3,var,bound));                                        
                     ;//  (*wmod)[fencode3_mpiu(*p,ii,var)]=(*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)];              
                     ;//  (*w)[fencode3_mpiu(*p,ii,var)]=(*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)];



                  }
         ;// printf("\n");
}*/





}




//copy mpi recv buffer to gpu memory     
int cucopywfrommpiw(struct params **p,real **w, real **wmod,    real **gmpiw0, real **gmpiwmod0,    real **gmpiw1, real **gmpiwmod1,    real **gmpiw2, real **gmpiwmod2, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw0, real **d_gmpiwmod0,   real **d_gmpiw1, real **d_gmpiwmod1,   real **d_gmpiw2, real **d_gmpiwmod2, int order, int idir)
{
       int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;     
       int szbuf;
     int szw0,szw1,szw2;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


  #ifdef USE_SAC
  
  szw0=4*NVAR*(  ((*p)->n[1])     );
  szw1=4*NVAR*(  ((*p)->n[0])     );

  #endif
  #ifdef USE_SAC_3D
  
   szw0=4*NVAR*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NVAR*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NVAR*(    ((*p)->n[0])*((*p)->n[1])  );

  #endif




      //copy data from w and wmod to correct gmpiw and gmpiwmod

//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
   /*  for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC_3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       (*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];  

                 //   if(var==4 && ((*p)->ipe)==1) 
                                 
                //        printf(" %d %d %d %d actual %d  mpi data%d %g %g\n",i1,i2,bound,dim,var,encodempiw0(*p,i1,i2,i3,var,bound),(*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)],  (*wmod)[fencode3_mpiu(*p,ii,var)] );
            
                       (*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];

                      //  if(var==4  && ((*p)->ipe)==1)
                     //  {
			//	(*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)]=0.5;
			//	(*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)]=0.5;
                      // }

                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC_3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;  

                       (*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];

                  }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                      (*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      

                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC_3D
                       case 2:
         i3=bound*(bound<2)+(((*p)->n[2])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 

                      (*gmpiwmod2)[encodempiw2(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw2)[encodempiw2(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      
                    }                            
                       break;                       
            #endif             
             }
                                     
         }    */

//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     /*for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC_3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;     

                      (*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      
  
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;

                      (*gmpiwmod0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw0)[encodempiw0(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC_3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 

                      (*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      
                   }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;


                      (*gmpiwmod1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw1)[encodempiw1(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC_3D
                       case 2:
         i3=bound*(bound<2)+(((*p)->n[2])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 


                      (*gmpiwmod2)[encodempiw2(*p,i1,i2,i3,var,bound)]=(*wmod)[fencode3_mpiu(*p,ii,var)];              
                       (*gmpiw2)[encodempiw2(*p,i1,i2,i3,var,bound)]=(*w)[fencode3_mpiu(*p,ii,var)];      
                   }                            
                       break;                       
            #endif             
             }
                                     
         }  */ 

#ifdef USE_GPUDIRECT
      mpiwtogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw0,*d_gmpiwmod0,*d_gmpiw1,*d_gmpiwmod1,*d_gmpiw2,*d_gmpiwmod2,idir);    
     hipDeviceSynchronize();

#else

if(idir==0)
{
   	 hipMemcpy(*d_gmpiw0, *gmpiw0, szw0*sizeof(real), hipMemcpyHostToDevice);     
   	 //hipMemcpy(*d_gmpiwmod0, *gmpiwmod0, szw0*sizeof(real), hipMemcpyHostToDevice); 
}

if(idir==1)
{
   	 hipMemcpy(*d_gmpiw1, *gmpiw1, szw1*sizeof(real), hipMemcpyHostToDevice);     
   	 //hipMemcpy(*d_gmpiwmod1, *gmpiwmod1, szw1*sizeof(real), hipMemcpyHostToDevice);     
}
    
            #ifdef USE_SAC_3D
if(idir==2)
{
   	      hipMemcpy(*d_gmpiw2, *gmpiw2, szw2*sizeof(real), hipMemcpyHostToDevice);     
   	     // hipMemcpy(*d_gmpiwmod2, *gmpiwmod0, szw2*sizeof(real), hipMemcpyHostToDevice); 
}    
         #endif

    //printf("call mpiwtogpu\n");

     mpiwtogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw0,*d_gmpiwmod0,*d_gmpiw1,*d_gmpiwmod1,*d_gmpiw2,*d_gmpiwmod2,idir);
     hipDeviceSynchronize();
     
     
 #endif    
     
}

int cucopywmodfrommpiw(struct params **p,real **w, real **wmod,    real **gmpiw0, real **gmpiwmod0,    real **gmpiw1, real **gmpiwmod1,    real **gmpiw2, real **gmpiwmod2, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw0, real **d_gmpiwmod0,   real **d_gmpiw1, real **d_gmpiwmod1,   real **d_gmpiw2, real **d_gmpiwmod2, int order, int idir)
{
       int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;     
       int szbuf;
     int szw0,szw1,szw2;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));



real *tgmpiwmod0=*gmpiwmod0;
real *tgmpiwmod1=*gmpiwmod1;


   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


  #ifdef USE_SAC
  
  szw0=4*NVAR*(  ((*p)->n[1])     );
  szw1=4*NVAR*(  ((*p)->n[0])     );

  #endif
  #ifdef USE_SAC_3D
  
   szw0=4*NVAR*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NVAR*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NVAR*(    ((*p)->n[0])*((*p)->n[1])  );

  #endif


#ifndef USE_GPUDIRECT
 
//printf("gpudirect not define!\n");

      //copy data from w and wmod to correct gmpiw and gmpiwmod




if(idir==0)
{



		    i3=0;
                   //   int bound,iside,n=0;
                     //for(iside=0;iside<2;iside++)
		    //for(int i1=0;i1<=1;i1++)
		      //for(int i2=0;i2<(*p)->n[1];i2++)
		      //{
			//iside=0;
                        // bound=i1+2*(iside>0);
			 
			//if(((*p)->ipe==0) /*&&  (*p)->it != -1     && iside==1 && (100*(p->ipe)+10*dim+iside)==101*/ )
			//{
                          //  printf("tini %d %d %d %lg  \n",bound,i2,i1,tgmpiwmod0[encodempiw0 (*p,i1, i2, i3, 0,bound)]);
                            // printf(" %d %d %d   \n",bound,i2,i1);

			//}
                   //   n++;
                    //}





   	 hipMemcpy(*d_gmpiwmod0, *gmpiwmod0, szw0*sizeof(real), hipMemcpyHostToDevice); 
}

if(idir==1)
{
   	 hipMemcpy(*d_gmpiwmod1, *gmpiwmod1, szw1*sizeof(real), hipMemcpyHostToDevice);     
}
    
            #ifdef USE_SAC_3D
if(idir==2)
{
   	      hipMemcpy(*d_gmpiwmod2, *gmpiwmod0, szw2*sizeof(real), hipMemcpyHostToDevice); 
}    
         #endif


  if(idir==1 /*&& (*p)->ipe==0     &&  idir==0*/ )
    {
        ;//printf("ipe2 mpiw0 after bound \n");
        
        ;//for(int j=0; j<4;j++)
         //for(int i=0; i<((*p)->n[1]);i++) 
        ;// for(int i=0; i<10;i++)              
         ;//    printf("%d %d %lg %lg\n",i,j, (tgmpiwmod0[4*rhob*((*p)->n[0]) +i+j*((*p)->n[0])]), (tgmpiwmod1[4*rhob*((*p)->n[0]) +i+j*((*p)->n[0])]));
         ;//printf("\n");
     }
     
 #endif



    //printf("call mpiwtogpu\n");

     mpiwmodtogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw0,*d_gmpiwmod0,*d_gmpiw1,*d_gmpiwmod1,*d_gmpiw2,*d_gmpiwmod2,idir,order);
     hipDeviceSynchronize();








}


//copy mpi recv buffer to gpu memory     
int cucopywdfrommpiwd(struct params **p,real **wd,     real **gmpiw0,     real **gmpiw1,     real **gmpiw2,  struct params **d_p  ,real **d_wd,    real **d_gmpiw0,   real **d_gmpiw1,   real **d_gmpiw2,  int order, int idir)
{
       int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;     
       int szbuf;
     int szw0,szw1,szw2;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


  #ifdef USE_SAC
  
  szw0=4*NDERV*(  ((*p)->n[1])     );
  szw1=4*NDERV*(  ((*p)->n[0])     );

  #endif
  #ifdef USE_SAC_3D
  
   szw0=4*NDERV*(  ((*p)->n[1])*((*p)->n[2])    );
  szw1=4*NDERV*(    ((*p)->n[0])*((*p)->n[2])   );
  szw2=4*NDERV*(    ((*p)->n[0])*((*p)->n[1])  );

  #endif


//#ifndef USE_GPUDIRECT


       if(idir==0)
   	 hipMemcpy(*d_gmpiw0, *gmpiw0, szw0*sizeof(real), hipMemcpyHostToDevice);     
 
	if(idir==1)
   	 hipMemcpy(*d_gmpiw1, *gmpiw1, szw1*sizeof(real), hipMemcpyHostToDevice);     

    
            #ifdef USE_SAC_3D
     if(idir==2)
   	      hipMemcpy(*d_gmpiw2, *gmpiw2, szw2*sizeof(real), hipMemcpyHostToDevice);     
         #endif
//#endif
    //printf("call mpiwtogpu\n");

     mpiwdtogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,0,0,*d_wd,*d_gmpiw0,*d_gmpiw1,*d_gmpiw2,idir);
     hipDeviceSynchronize();
}



//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopytompivisc(struct params **p,real **temp2, real **gmpivisc0, real **gmpivisc1, real **gmpivisc2,  struct params **d_p,real **d_wtemp2,    real **d_gmpivisc0,    real **d_gmpivisc1,    real **d_gmpivisc2)
{


     int szbuf,szbuf0,szbuf1,szbuf2;
     int dim,bound,var=0;
     int i1,i2,i3;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
             int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
	  szbuf0=4*(  (((*p)->n[1])+2 )   );
	  szbuf1=4*(    (((*p)->n[0]) +2 )  );


     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
  szbuf0=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)  ); 
  szbuf1=4*(   (((*p)->n[0])+2)*(((*p)->n[2])+2)    );    
  szbuf2=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)   );    


     #endif
     gputompivisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wtemp2,*d_gmpivisc0,*d_gmpivisc1,*d_gmpivisc2);
     hipDeviceSynchronize();
     hipMemcpy(*gmpivisc0, *d_gmpivisc0, szbuf0*sizeof(real), hipMemcpyDeviceToHost);
     hipMemcpy(*gmpivisc1, *d_gmpivisc1, szbuf1*sizeof(real), hipMemcpyDeviceToHost);
     #ifdef USE_SAC_3D
     	hipMemcpy(*gmpivisc2, *d_gmpivisc2, szbuf2*sizeof(real), hipMemcpyDeviceToHost);
     #endif
     //copy data to correct area in temp2
//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     /*  for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<2;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC_3D
         i1=bound*(((*p)->n[0])+1);
         for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {     
                        
          //i1=(p->n[0])+1;
         
          //temp2[encode3p2_sacmpi (p,i1, i2, i3, tmpnui)]=gmpitgtbufferr[0][i2+i3*((p->n[1])+2)];
          //temp2[encode3p2_sacmpi (p,0, i2, i3, tmpnui)]=gmpitgtbufferl[0][i2+i3*((p->n[1])+2)];
         
                       (*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)]=(*gmpivisc0)[encodempivisc0(*p,i1,i2,i3,bound,dim)];
                  }
            #else
         i3=0;
         i1=bound*(((*p)->n[0])+1);
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                       (*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)]=(*gmpivisc0)[encodempivisc0(*p,i1,i2,i3,bound,dim)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC_3D
         i2=bound*(((*p)->n[1])+1);
         for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {
                       (*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)]=(*gmpivisc1)[encodempivisc1(*p,i1,i2,i3,bound,dim)];
                  }

            #else
         i3=0;
         i2=bound*(((*p)->n[1])+1);
                  for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  {                                                       
                       (*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)]=(*gmpivisc1)[encodempivisc1(*p,i1,i2,i3,bound,dim)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC_3D
                       case 2:
                  i3=bound*(((*p)->n[2])+1);
        for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {                                                           
                       (*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)]=(*gmpivisc2)[encodempivisc2(*p,i1,i2,i3,bound,dim)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    */

}

//copy mpi recv buffer to gpu memory     
int cucopyfrommpivisc(struct params **p,real **temp2,real **gmpivisc0,real **gmpivisc1,real **gmpivisc2,  struct params **d_p,real **d_wtemp2,    real **d_gmpivisc0,    real **d_gmpivisc1,    real **d_gmpivisc2)
{
      int dim,bound,var=0;
     int i1,i2,i3;      
 
     int szbuf,szbuf0,szbuf1,szbuf2;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     
 	  szbuf0=4*(  (((*p)->n[1])+2 )   );
	  szbuf1=4*(    (((*p)->n[0]) +2 )  );

     #ifdef USE_SAC_3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );

     
  szbuf0=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)  ); 
  szbuf1=4*(   (((*p)->n[0])+2)*(((*p)->n[2])+2)    );    
  szbuf2=4*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)   ); 

     #endif

      //copy data from temp2 to gmpivisc
        /*     for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<2;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC_3D
         i1=bound*(((*p)->n[0])+1);
         for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {     
                        
          //i1=(p->n[0])+1;
         
          //temp2[encode3p2_sacmpi (p,i1, i2, i3, tmpnui)]=gmpitgtbufferr[0][i2+i3*((p->n[1])+2)];
          //temp2[encode3p2_sacmpi (p,0, i2, i3, tmpnui)]=gmpitgtbufferl[0][i2+i3*((p->n[1])+2)];
         
                       (*gmpivisc0)[encodempivisc0(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)];
                  }
            #else
         i3=0;
         i1=bound*(((*p)->n[0])+1);
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                       (*gmpivisc0)[encodempivisc0(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC_3D
         i2=bound*(((*p)->n[1])+1);
         for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {
                       (*gmpivisc1)[encodempivisc1(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)];
                  }

            #else
         i3=0;
         i2=bound*(((*p)->n[1])+1);
                  for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  {
                                                                     
                       (*gmpivisc1)[encodempivisc1(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC_3D
                       case 2:
                  i3=bound*(((*p)->n[2])+1);
        for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                                                              
                       (*gmpivisc2)[encodempivisc2(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_mpiu(*p,i1,i2,i3,var)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         } */   


   	 hipMemcpy(*d_gmpivisc0, *gmpivisc0, szbuf0*sizeof(real), hipMemcpyHostToDevice);     
   	 hipMemcpy(*d_gmpivisc1, *gmpivisc1, szbuf1*sizeof(real), hipMemcpyHostToDevice);
       #ifdef USE_SAC_3D    
   	 hipMemcpy(*d_gmpivisc2, *gmpivisc2, szbuf2*sizeof(real), hipMemcpyHostToDevice); 
       #endif    

     mpivisctogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wtemp2,*d_gmpivisc0,*d_gmpivisc1,*d_gmpivisc2);
     hipDeviceSynchronize();
}


#endif



