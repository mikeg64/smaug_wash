#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,xc3;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa;
   real s_period;
   real qt,tdep,tdepx,tdepy;

   real vx,vy,vz;

   real exp_x,exp_y,exp_z,exp_xyz;


   real xp,yp,zp;
   int i,j,k;
 	  
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

     xc1=0.1e6;
    xc2=1.0e6;
    xc3=1.0e6;

          xp=(p->xmin[1])+(((real)j)*(p->dx[1]))-xc2;
          zp=(p->xmin[0])+(((real)i)*(p->dx[0]))-xc1;
          yp=(p->xmin[2])+(((real)k)*(p->dx[2]))-xc3;
     // xx=x(ix_1,ix_2,ix_3,2)-xc2
     // yy=x(ix_1,ix_2,ix_3,3)-xc3
     // zz=x(ix_1,ix_2,ix_3,1)-xc1  


          xp=wd[fencode3_MODID(p,ii,pos2)]-xc2;
          zp=wd[fencode3_MODID(p,ii,pos1)]-xc1;
          yp=wd[fencode3_MODID(p,ii,pos3)]-xc3;  
     

    xxmax=2.0e6;
    yymax=2.0e6;

    dx=0.1e6;
    dy=0.1e6;
    dz=0.05e6;

    aa=20;
    s_period=180.0;
    tdep=1.00;
    qt=p->qt;

        //exp_z=exp(-zz**2.d0/(delta_z**2.d0))
        //exp_x=exp(-xx**2.d0/(delta_x**2.d0))
        //exp_y=exp(-yy**2.d0/(delta_y**2.d0))       
        //exp_xyz=exp_x*exp_y*exp_z
        exp_z=exp(-zp*zp/(dz*dz));
        exp_x=exp(-xp*xp/(dx*dx));
        exp_y=exp(-yp*yp/(dy*dy));       
        exp_xyz=exp_x*exp_y*exp_z;

        //vvx(ix_1,ix_2,ix_3)=AA*yy/yymax*exp_xyz*tdep    
        //vvy(ix_1,ix_2,ix_3)=-AA*xx/xxmax*exp_xyz*tdep 

        //torsional driver
        //vx=(aa*yp/yymax)*exp_xyz*tdep;    
        //vy=-(aa*xp/xxmax)*exp_xyz*tdep;


       //washing machine driver
        tdepx=sin(qt*2.0*PI/s_period);
        tdepy=sin((qt-(0.25*s_period))*2.0*PI/s_period);
        aa=aa*exp(-(qt-582)/90);
        vx=aa*exp_xyz*tdepx;    
        vy=aa*exp_xyz*tdepy;




 

                           w[fencode3_MODID(p,ii,mom2)]+=(p->dt)*vx*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
  
                           w[fencode3_MODID(p,ii,mom3)]+=(p->dt)*vy*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);

                          w[fencode3_MODID(p,ii,energy)]+=(p->dt)*(vx*vx+vy*vy)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;

        //if(i==5 && j==63 && k==63) 
	//{
               // p->test=(w[fencode3_MODID(p,ii,energy)]);
               // p->chyp[0]=(w[fencode3_MODID(p,ii,mom2)]);
               // p->chyp[1]=(w[fencode3_MODID(p,ii,mom3)]);


             //  printf("%d %d %d %f %f %f %f %f %f\n",i,j,k,qt,tdepx,tdepy,exp_xyz,vx,vy); 
	//}

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {


}

